#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../gptl.h"
#include "../cuda/gptl_cuda.h"


__global__ void runit (float, float);

int main ()
{
  static int blocksize = 128;
  int warpsize = -1;
  int khz = -1;
  int devnum = -1;
  int smcount = -1;
  int cores_per_sm = -1;
  int cores_per_gpu = -1;
  int oversub = -1;
  int nwarps;
  int nthreads;
  int nblocks;
  int ans;
  int ok;
  
  int ret;
  float sleep_tot;
  float sleep_percall;

  ret = GPTLget_gpu_props (&khz, &warpsize, &devnum, &smcount, &cores_per_sm, &cores_per_gpu);
  printf ("warpsize=%d\n",      warpsize);
  printf ("smcount=%d\n",       smcount);
  printf ("cores_per_sm=%d\n",  cores_per_sm);
  printf ("cores_per_gpu=%d\n", cores_per_gpu);

  printf ("Enter oversubsubscription factor\n");
  scanf ("%d", &oversub);
  printf ("oversub=%d\n", oversub);

  nwarps = (cores_per_gpu * oversub) / warpsize;
  printf ("nwarps=%d\n", nwarps);
  if (nwarps * warpsize != cores_per_gpu * oversub)
    printf ("NOTE: warpsize=%d does not divide evenly into cores_per_gpu(%d) * oversub(%d)=%d\n",
	    warpsize, cores_per_gpu, oversub, cores_per_gpu * oversub);
  ret = GPTLsetoption (GPTLmaxwarps_gpu, nwarps);

  printf ("Enter sleep_tot sleep_percall (both in floating point seconds)\n");
  scanf ("%f%f", &sleep_tot, &sleep_percall);
  printf ("sleep_tot=%f sec sleep_percall=%f sec\n", sleep_tot, sleep_percall);
  
  ret = GPTLinitialize ();
  ret = GPTLstart ("total");
  nthreads = nwarps * warpsize;
  nblocks = nthreads / blocksize;
  printf ("nblocks=%d blocksize=%d\n", nblocks, blocksize);
  runit <<<nblocks,blocksize>>> (sleep_tot, sleep_percall);
  hipDeviceSynchronize ();
  ret = GPTLstop ("total");
  ret = GPTLpr (0);
  return 0;
}

__global__ void runit (float sleep_tot, float sleep_percall)
{
  int ret;
  double slept = 0.;
  __shared__ double accum;
  __shared__ double maxtime, mintime;

  ret = GPTLstart_gpu ("runit");
  while (slept < sleep_tot) {
    ret = GPTLstart_gpu ("percall");
    ret = GPTLmy_sleep (sleep_percall);
    ret = GPTLstop_gpu ("percall");
    if (true) {
      slept += sleep_percall;
    } else if (threadIdx.x == 0) {
      ret = GPTLget_wallclock_gpu ("percall", &accum, &maxtime, &mintime);
      __syncthreads();
      slept += accum;
      printf ("threadIdx.x=%d slept=%f\n", threadIdx.x, slept);
    }
  }
  ret = GPTLstop_gpu ("runit");
}
