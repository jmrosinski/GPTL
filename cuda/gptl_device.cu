#include "hip/hip_runtime.h"
/*
** gptl.cu
** Author: Jim Rosinski
**
** Main file contains most CUDA GPTL functions
*/

#include <stdio.h>
#include <string.h>        // memcpy
#include <stdint.h>        // uint types
#include <hip/hip_runtime.h>

#include "./private.h"
#include "./gptl_cuda.h"

#define FLATTEN_TIMERS(SUB1,SUB2) (SUB1)*maxtimers + (SUB2)

__device__ static Timer *timers = 0;            // array (also linked list) of timers
__device__ static Timername *timernames;        // array of timer names
__device__ static int max_name_len;             // max length of timer name
__device__ static int ntimers = 0;              // number of timers
__device__ __constant__ static int maxtimers;   // max number of timers
__device__ static int maxwarps = -1;            // max warps
__device__ static int maxwarpid_found = 0;      // number of warps found : init to 0
__device__ static int maxwarpid_timed = 0;      // number of warps analyzed : init to 0
__device__ static bool initialized = false;     // GPTLinitialize has been called
__device__ static bool verbose = false;         // output verbosity
__device__ static double gpu_hz = 0.;           // clock freq
__device__ static volatile int mutex = 0;       // critical section unscrambles printf output

extern "C" {

// Local function prototypes
__global__ static void initialize_gpu (const int, const int, const double, Timer *, Timername *);
__device__ static inline int get_warp_num (void);         // get 0-based 1d warp number
__device__ static inline int update_stats_gpu (const int, Timer *, const long long, const int,
					       const uint);
__device__ static int my_strlen (const char *);
__device__ static char *my_strcpy (char *, const char *);
__device__ static int my_strcmp (const char *, const char *);
__device__ static void start_misc (int, const int);
__device__ static void stop_misc (int w, const int handle);
__device__ static void init_gpustats (Gpustats *, int);
__device__ static void fill_gpustats (Gpustats *, int, int);
// Defining PRINTNEG will print to stdout whenever a negative interval (stop minus start) is
// encountered. Only useful when non-zero negative intervals are reported in timing output
#undef PRINTNEG
#ifdef PRINTNEG
__device__ static void prbits8 (uint64_t);
#endif

/* VERBOSE is a debugging ifdef local to the rest of this file */
#define VERBOSE

__host__ int GPTLinitialize_gpu (const int verbose_in,
				 const int maxwarps_in,
				 const int maxtimers_in,
				 const double gpu_hz_in)
{
  size_t nbytes;  // number of bytes to allocate
  static Timer *timers_cpu = 0;          // array of timers
  static Timername *timernames_cpu = 0; // array of timer names

  nbytes = maxwarps_in * maxtimers_in * sizeof (Timer);
  gpuErrchk (hipMalloc (&timers_cpu, nbytes));

  nbytes =               maxtimers_in * sizeof (Timername);
  gpuErrchk (hipMalloc (&timernames_cpu, nbytes));

  // Set constant memory values: First arg is pass by reference so no "&"
  gpuErrchk (hipMemcpyToSymbol(HIP_SYMBOL(maxtimers),   &maxtimers_in,    sizeof (int)));

  initialize_gpu <<<1,1>>> (verbose_in,
			    maxwarps_in,
			    gpu_hz_in,
			    timers_cpu,
			    timernames_cpu);
  // This should flush any existing print buffers
  hipDeviceSynchronize ();
  return 0;
}

/*
** GPTLinitialize_gpu (): Initialization routine must be called from single-threaded
**   region before any other timing routines may be called.  The need for this
**   routine could be eliminated if not targetting timing library for threaded
**   capability. 
*/
__global__ static void initialize_gpu (const int verbose_in,
				       const int maxwarps_in,
				       const double gpu_hz_in,
				       Timer *timers_cpu,
				       Timername *timernames_cpu)
{
  int w, wi;        // warp, flattened indices
  long long t1, t2; // returned from underlying timer
  static const char *thisfunc = "initialize_gpu";

#ifdef VERBOSE
  printf ("Entered %s\n", thisfunc);
#endif
  if (initialized) {
    (void) GPTLerror_1s ("%s: has already been called\n", thisfunc);
    return;
  }

  // Set global vars from input args
  verbose           = verbose_in;
  maxwarps          = maxwarps_in;
  gpu_hz            = gpu_hz_in;
  timers            = timers_cpu;
  timernames        = timernames_cpu;

  // Initialize timers
  ntimers = 0;
  max_name_len = 0;
  for (w = 0; w < maxwarps; ++w) {
    wi = FLATTEN_TIMERS(w,0);
    memset (&timers[wi], 0, maxtimers * sizeof (Timer));
  }
  // Make a timer "GPTL_ROOT" to ensure no orphans, and to simplify printing.
  memcpy (timernames[0].name, "GPTL_ROOT", 9+1);

  if (verbose) {
    t1 = clock64 ();
    t2 = clock64 ();
    if (t1 > t2)
      printf ("GPTL %s: negative delta-t=%lld\n", thisfunc, t2-t1);

    printf ("Per call overhead est. t2-t1=%g should be near zero\n", t2-t1);
    printf ("Underlying wallclock timing routine is clock64\n");
  }

  initialized = true;
  printf("end %s: maxwarps=%d\n", thisfunc, maxwarps);
}

/*
** GPTLfinalize_gpu (): Finalization routine must be called from single-threaded
**   region. Free all malloc'd space
*/
__global__ void GPTLfinalize_gpu (void)
{
  static const char *thisfunc = "GPTLfinalize_gpu";

  if ( ! initialized) {
    (void) GPTLerror_1s ("%s: initialization was not completed\n", thisfunc);
    return;
  }

  hipFree (timers);
  hipFree (timernames);
  
  GPTLreset_errors_gpu ();

  // Reset initial values
  timers = 0;
  timernames = 0;
  max_name_len = 0;
  initialized = false;
  verbose = false;
}

/*
** GPTLinit_handle_gpu: Initialize a handle for further use by GPTLstart_gpu() and GPTLstop_gpu()
**
** Input arguments:
**   name: timer name
**
** Output arguments:
**   handle: Index into array for "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLinit_handle_gpu (const char *name,     // timer name
				    int *handle)          // handle
{
  int numchars;      // length of "name"
  int mywarp;        // my warp number
  int i;
  static const char *thisfunc = "GPTLinit_handle_gpu";

  // Guts of this function are run only by thread 0 of warp 0 due to loop over warps below. 
  // Need to have each timer have the same index for all warps.
  // Nice feature: Can be called by just thread 0 of warp 0, OR NOT
  if ((mywarp = get_warp_num ()) != 0) {
    return SUCCESS;
  }

  // First check if a handle for the requested timer already exists (i=1 skips GPTL_ROOT)
  for (i = 1; i <= ntimers; ++i) {
    if (STRMATCH (name, timernames[i].name)) {
#ifdef DEBUG_PRINT
      printf ("%s name=%s: Returning already existing handle=%d\n", thisfunc, name, i);
#endif
      *handle = i;
      return SUCCESS;
    }
  }
  
  if (ntimers >= maxtimers) {
    return GPTLerror_2s1d ("%s: Too many timers. name=%s maxtimers needs to be incremented from %d\n",
			   thisfunc, name, maxtimers);
  } else {
    numchars = MIN (my_strlen (name), MAX_CHARS);
    max_name_len = MAX (numchars, max_name_len);
    *handle = ++ntimers;
    memcpy (timernames[ntimers].name, name, numchars);
    timernames[ntimers].name[numchars] = '\0';
  }
  //  printf ("%s name=%s: mywarp=%d Returning new handle=%d\n", thisfunc, name, mywarp, *handle);
      
  return SUCCESS;
}

/*
** GPTLstart_gpu: start a timer based on a handle
**
** Input arguments:
**   name: timer name (required when on input, handle=0)
**   handle: pointer to timer matching "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLstart_gpu (const int handle)
{
  Timer *ptr;        // linked list pointer
  int w;             // warp index (of this thread)
  int wi;            // flattened 2d index for warp number and timer name

  static const char *thisfunc = "GPTLstart_gpu";

  if ( ! initialized)
    return GPTLerror_1s1d ("%s handle=%d: GPTLinitialize_gpu has not been called\n", 
			   thisfunc, handle);

  w = get_warp_num ();

  // Return if not thread 0 of the warp, or warpId is outside range of available timers
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    return SUCCESS;

  // Input handle should be a positive integer not greater than ntimers (0 accepted for GPTL_ROOT)
  if (handle < 0 || handle > ntimers)
    return GPTLerror_1s1d ("%s: Invalid input handle=%d. Perhaps GPTLinit_handle_gpu not called?\n",
			   thisfunc, handle);

  wi = FLATTEN_TIMERS (w, handle);
  ptr = &timers[wi];
  
  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr->onflg) {
    ++ptr->recurselvl;
    return SUCCESS;
  }

#ifdef DEBUG_PRINT
  printf ("%s: ptr=%p setting onflg=true\n", thisfunc, ptr);
#endif

  // Get the timestamp and smid
  // IMPORTANT: Issue the cmds in sequence because SM changing between clock64() call and getting
  // the SM number would be bad.
  // NOTE: Timing value will be thrown away if SM changes upon stop() call.
  ptr->wall.last = clock64 ();
  asm volatile ("mov.u32 %0, %smid;" : "=r"(ptr->smid));
  ptr->onflg = true;

  return SUCCESS;
}

/*
** GPTLstop_gpu: stop a timer based on a handle
**
** Input arguments:
**   name: timer name (used only for diagnostics)
**   handle: pointer to timer
**
** Return value: 0 (success) or -1 (failure)
*/
__device__ int GPTLstop_gpu (const int handle)
{
  register long long tp1;    // time stamp
  Timer timer;               // local copy of timers[wi]: gives some speedup vs. global array
  int w;                     // warp number for this process
  int wi;                    // flattened (1-d) index into 2-d array [timer][warp]
  uint smid;                 // SM id
  static const char *thisfunc = "GPTLstop_gpu";

  if ( ! initialized)
    return GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);

  w = get_warp_num ();

  // Return if not thread 0 of the warp, or warpId is outside range of available timers
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    return SUCCESS;

  // Input handle should be a positive integer not greater than ntimers (0 accepted for GPTL_ROOT)
  if (handle < 0 || handle > ntimers)
    return GPTLerror_1s1d ("%s: Invalid input handle=%d. Perhaps GPTLinit_handle_gpu not called?\n",
			   thisfunc, handle);

  // Get the timestamp and smid
  // IMPORTANT: Issue the cmds in sequence because SM changing between clock64() call and getting
  // the SM number would be bad.
  // NOTE: Timing value will be thrown away if SM changed from start() call.
  tp1 = clock64 ();
  asm ("mov.u32 %0, %smid;" : "=r"(smid));

  wi = FLATTEN_TIMERS (w, handle);
  timer = timers[wi];

  if ( ! timer.onflg )
    return GPTLerror_2s ("%s: timer %s was already off.\n", thisfunc, timernames[handle].name);

  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (timer.recurselvl > 0) {
    --timer.recurselvl;
    ++timer.count;
    timers[wi] = timer;
    return SUCCESS;
  }

  if (update_stats_gpu (handle, &timer, tp1, w, smid) != 0)
    return GPTLerror_1s ("%s: error from update_stats_gpu\n", thisfunc);
#ifdef DEBUG_PRINT
  printf ("%s: handle=%d count=%d\n", thisfunc, handle, (int) timer.count);
#endif
  timers[wi] = timer;
  
  return SUCCESS;
}

/*
** update_stats_gpu: update stats inside ptr. Called by GPTLstop_gpu, GPTLstop_handle_gpu
**
** Input arguments:
**   ptr: pointer to timer
**   tp1: input time stamp
**   w: warp index
**
** Return value: 0 (success) or GPTLerror (failure)
*/

__device__ static inline int update_stats_gpu (const int handle,
					       Timer *ptr, 
					       const long long tp1, 
					       const int w,
					       const uint smid)
{
  register long long delta;           // time diff from start()
  static const char *thisfunc = "update_stats_gpu";
#ifdef DEBUG_PRINT
  printf ("%s: ptr=%p setting onflg=false\n", thisfunc, ptr);
#endif

  ptr->onflg = false;
  delta = tp1 - ptr->wall.last;

  if (smid != ptr->smid) {
#ifdef DEBUG_PRINT
    printf ("GPTL %s: name=%s w=%d sm changed from %d to %d: SKIPPING timing\n", 
	    thisfunc, timernames[handle].name, w, ptr->smid, smid);
#endif
    ++ptr->badsmid_count;
    return SUCCESS;
  }

  if (delta < 0) {
#ifdef PRINTNEG
    bool isSet; 
    // Use critical section so printf from multiple SMs don't get scrambled
    do {
      // If mutex is 0, grab by setting = 1
      // If mutex is 1, it stays 1 and isSet will be false
      isSet = atomicCAS ((int *) &mutex, 0, 1) == 0; 
      if (isSet) {  // critical section starts here
	printf ("GPTL: %s name=%s w=%d WARNING NEGATIVE DELTA ENCOUNTERED: %lld-%lld=%lld=%g seconds: IGNORING\n", 
		thisfunc, timernames[handle].name, w, tp1, ptr->wall.last, delta, delta / (-gpu_hz));
	printf ("Bit pattern old:");
	prbits8 ((uint64_t) ptr->wall.last);

	printf ("Bit pattern new:");
	prbits8 ((uint64_t) tp1);
	mutex = 0;     // end critical section by releasing the mutex
      }
    } while ( !isSet); // exit the loop after critical section executed
#endif
    
    ++ptr->negdelta_count;
    return SUCCESS;
  }

  ++ptr->count;
  ptr->wall.accum += delta;

  if (ptr->count == 1) {
    ptr->wall.max = delta;
    ptr->wall.min = delta;
  } else {
    if (delta > ptr->wall.max)
      ptr->wall.max = delta;
    if (delta < ptr->wall.min)
      ptr->wall.min = delta;
  }

  return SUCCESS;
}

/*
** GPTLreset_gpu: reset all timers to 0
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__global__ void GPTLreset_gpu (void)
{
  int i;
  int w;             /* index over warps */
  int wi;
  static const char *thisfunc = "GPTLreset_gpu";

  if ( ! initialized) {
    (void) GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);
    return;
  }

  for (w = 0; w <= maxwarpid_timed; ++w) {
    for (i = 0; i < maxtimers; ++i) {
      wi = FLATTEN_TIMERS(w,i);
      timers[wi].onflg = false;
      timers[wi].count = 0;
      memset (&timers[wi].wall, 0, sizeof (timers[wi].wall));
    }
  }

  if (verbose)
    printf ("%s: accumulators for all GPU timers set to zero\n", thisfunc);
}

__device__ static inline int get_warp_num ()
{
  int threadId;
  int warpId;

  threadId = threadIdx.x
        +  blockDim.x  * threadIdx.y
        +  blockDim.x  *  blockDim.y  * threadIdx.z
        +  blockDim.x  *  blockDim.y  *  blockDim.z  * blockIdx.x
        +  blockDim.x  *  blockDim.y  *  blockDim.z  *  gridDim.x  * blockIdx.y
        +  blockDim.x  *  blockDim.y  *  blockDim.z  *  gridDim.x  *  gridDim.y  * blockIdx.z;

  // Only thread 0 of the warp will be timed
  if (threadId % WARPSIZE != 0)
    return NOT_ROOT_OF_WARP;

  warpId = threadId / WARPSIZE;

  // maxwarpid_found is needed only by CPU code when printing results
  if (warpId+1 > maxwarpid_found)
    maxwarpid_found = warpId;

  if (warpId > maxwarps-1)
    return WARPID_GT_MAXWARPS;

  // if we get here we have a usable warpId
  if (warpId > maxwarpid_timed)
    maxwarpid_timed = warpId;

  return warpId;
}

__global__ void GPTLget_gpusizes (int *maxwarpid_found_out, int *maxwarpid_timed_out)
{
  *maxwarpid_found_out = maxwarpid_found;
  *maxwarpid_timed_out = maxwarpid_timed;
}

__device__ int GPTLget_wallclock_gpu (const int handle,
				      double *accum, double *max, double *min)
{
  int w;               // warp index
  int wi;
  static const char *thisfunc = "GPTLget_wallclock_gpu";
  
  if ( ! initialized)
    (void) GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);

  if (gpu_hz == 0.)
    (void) GPTLerror_1s ("%s: gpu_hz has not been set\n", thisfunc);

  w = get_warp_num ();
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    return SUCCESS;

  if (handle < 0 || handle > ntimers)
    return GPTLerror_1s1d ("%s: bad handle=%d\n", thisfunc, handle);

  wi = FLATTEN_TIMERS (w, handle);
  
  *accum = timers[wi].wall.accum / gpu_hz;
  *max   = timers[wi].wall.max   / gpu_hz;
  *min   = timers[wi].wall.min   / gpu_hz;
  return 0;
}

//JR want to use variables to dimension arrays but nvcc is not C99 compliant
__global__ void GPTLfill_gpustats (Gpustats *gpustats, 
				   int *max_name_len_out,
				   int *ngputimers)
{
  int w;      // warp index
  int n;      // timer index

  *max_name_len_out = max_name_len;
  *ngputimers = ntimers;

  // Step 1: process entries for all warps based on those in warp 0
  // gpustats starts at 0. timers start at 1
  for (n = 0; n < ntimers; ++n) {
    init_gpustats (&gpustats[n], n+1);
    for (w = 1; w <= maxwarpid_timed; ++w) {
      fill_gpustats (&gpustats[n], n+1, w);
    }
  }

#ifdef DEBUG_PRINT
  printf ("%s: ngputimers=%d\n", thisfunc, n);
  for (n = 0; n < *ngputimers; ++n) {
    printf ("%s: timer=%s accum_max=%lld accum_min=%lld count_max=%d nwarps=%d\n", 
	    thisfunc, gpustats[n].name, gpustats[n].accum_max, gpustats[n].accum_min, gpustats[n].count_max, gpustats[n].nwarps);
  }
#endif
  return;
}

__device__ static void init_gpustats (Gpustats *gpustats, int idx)
{
  const int w = 0;
  (void) my_strcpy (gpustats->name, timernames[idx].name);
  gpustats->count  = timers[idx].count;
  if (timers[idx].count > 0)
    gpustats->nwarps = 1;
  else
    gpustats->nwarps = 0;

  gpustats->accum_max      = timers[idx].wall.accum;
  gpustats->accum_max_warp = w;

  gpustats->accum_min      = timers[idx].wall.accum;
  gpustats->accum_min_warp = w;

  gpustats->count_max      = timers[idx].count;
  gpustats->count_max_warp = w;

  gpustats->count_min      = timers[idx].count;
  gpustats->count_min_warp = w;

  gpustats->negdelta_count_max       = timers[idx].negdelta_count;
  gpustats->negdelta_count_max_warp  = w;
  gpustats->negdelta_nwarps          = timers[idx].negdelta_count  > 0 ? 1 : 0;

  gpustats->badsmid_count  = timers[idx].badsmid_count;
}

__device__ static void fill_gpustats (Gpustats *gpustats, int idx, int w)
{
  int wi = FLATTEN_TIMERS (w,idx);
  
  if (timers[wi].count > 0) {
    gpustats->count += timers[wi].count;
    ++gpustats->nwarps;

    if (timers[wi].wall.accum > gpustats->accum_max) {
      gpustats->accum_max      = timers[wi].wall.accum;
      gpustats->accum_max_warp = w;
    }
    
    if (timers[wi].wall.accum < gpustats->accum_min) {
      gpustats->accum_min      = timers[wi].wall.accum;
      gpustats->accum_min_warp = w;
    }
    
    if (timers[wi].count > gpustats->count_max) {
      gpustats->count_max      = timers[wi].count;
      gpustats->count_max_warp = w;
    }
    
    if (timers[wi].count < gpustats->count_min) {
      gpustats->count_min      = timers[wi].count;
      gpustats->count_min_warp = w;
    }
    
    if (timers[wi].negdelta_count > gpustats->negdelta_count_max) {
      gpustats->negdelta_count_max      = timers[wi].negdelta_count;
      gpustats->negdelta_count_max_warp = w;
    }

    if (timers[wi].negdelta_count > 0)
      ++gpustats->negdelta_nwarps;

    gpustats->badsmid_count += timers[wi].badsmid_count;
  }
}

__device__ static int my_strlen (const char *str)
{
  const char *s;
  for (s = str; *s; ++s);
  return(s - str);
}

__device__ static inline char *my_strcpy (char *dest, const char *src)
{
  char *ret = dest;

  while (*src != '\0')
    *dest++ = *src++;
  *dest = '\0';
  return ret;
}

//JR Both of these have about the same performance
__device__ static int my_strcmp (const char *str1, const char *str2)
{
#ifndef MINE
  while (*str1 == *str2) {
    if (*str1 == '\0')
      break;
    ++str1;
    ++str2;
  }
  return (int) (*str1 - *str2);
#else
  register const unsigned char *s1 = (const unsigned char *) str1;
  register const unsigned char *s2 = (const unsigned char *) str2;
  register unsigned char c1, c2;
 
  do {
      c1 = (unsigned char) *s1++;
      c2 = (unsigned char) *s2++;
      if (c1 == '\0')
	return c1 - c2;
  } while (c1 == c2); 
  return c1 - c2;
#endif
}

// Overhead estimate functions start here
/*
** GPTLget_overhead: return current status info about a timer. If certain stats are not enabled, 
** they should just have zeros in them.
** 
** Output args:
**   get_warp_num_ohd: Getting my warp index
**   utr_ohd:            Underlying timer routine
**   self_ohd:           Estimate of GPTL-induced overhead in the timer itself (included in "Wallclock")
**   parent_ohd:         Estimate of GPTL-induced overhead for the timer which appears in its parents
*/
__global__ void GPTLget_overhead_gpu (long long *get_warp_num_ohd,  // Getting my warp index
				      long long *startstop_ohd,     // start/stop pair
				      long long *utr_ohd,           // Underlying timing routine
				      long long *start_misc_ohd,    // misc start code
				      long long *stop_misc_ohd,     // misc stop code
				      long long *self_ohd,          // OHD in timer itself
				      long long *parent_ohd,        // OHD in parent
				      long long *my_strlen_ohd,
				      long long *STRMATCH_ohd)
{
  volatile uint smid;         // SM id
  long long t1, t2;           // Initial, final timer values
  int i;
  int ret;
  int mywarp;                 // our warp number
  char name[MAX_CHARS+1];     // Name to be used for various OHD tests
  char samename[MAX_CHARS+1]; // Copy of "name" for STRMATCH test

  // Define name to be used in OHD estimates. Use GPTL_ROOT because it's always there
  my_strcpy (name, timernames[0].name); // GPTL_ROOT
  my_strcpy (samename, name);

  /*
  ** Gather timings by running each test 1000 times
  ** First: start/stop overhead 
  */
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    ret = GPTLstart_gpu (0);
    ret = GPTLstop_gpu (0);
  }
  t2 = clock64();
  startstop_ohd[0] = (t2 - t1) / 1000;

  // get_warp_num overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    mywarp = get_warp_num ();
  }
  t2 = clock64();
  get_warp_num_ohd[0] = (t2 - t1) / 1000;

  // utr overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    asm volatile ("mov.u32 %0, %smid;" : "=r"(smid));
    t2 = clock64();
  }
  *utr_ohd = (t2 - t1) / 1000;

  // start misc overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    start_misc (0, 0);  // w, handle (handle=0 is GPTL_ROOT)
  }
  t2 = clock64();
  start_misc_ohd[0] = (t2 - t1) / 1000;

  // stop misc overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    stop_misc (0, 0);  // w, handle (handle=0 is GPTL_ROOT)
  }
  t2 = clock64();
  stop_misc_ohd[0] = (t2 - t1) / 1000;

  // Self and parent OHD estimates: A few settings at the end of GPTLstart_gpu should instead be 
  // applied to parent. A few settings at the beginning of GPTLstop_gpu should instead be
  // applied to self. But those errors are likely minor.
  self_ohd[0]   = utr_ohd[0] + start_misc_ohd[0];
  parent_ohd[0] = utr_ohd[0] + 2*get_warp_num_ohd[0] + stop_misc_ohd[0];

  // my_strlen overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    ret = my_strlen (name);
  }
  t2 = clock64();
  *my_strlen_ohd = (t2 - t1) / 1000;

  // STRMATCH overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    ret = STRMATCH (samename, name);
  }
  t2 = clock64();
  *STRMATCH_ohd = (t2 - t1) / 1000;
  return;
}

__device__ static void start_misc (int w, const int handle)
{
  int wi;
  Timer *ptr;
  static const char *thisfunc = "startmisc";

  if ( ! initialized)
    printf ("%s: ! initialized\n", thisfunc);

  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    printf ("%s: bad w value\n", thisfunc);

  if (handle < 0 || handle > ntimers)
    printf ("%s: bad handle value\n", handle);

  wi = FLATTEN_TIMERS (w, handle);
  ptr = &timers[wi];

  if (ptr->onflg) {
    ++ptr->recurselvl;
    printf ("%s: onflg should be off\n", thisfunc);
    ptr->smid = 0;
    ptr->wall.last = 0L;
  }
  ptr->onflg = false;  // GPTLstart actually sets this true but set false for better OHD est.
}

__device__ static void stop_misc (int w, const int handle)
{
  int wi;
  Timer timer;
  static const char *thisfunc = "stopmisc";

  if ( ! initialized)
    printf ("%s: ! initialized\n", thisfunc);

  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    printf ("%s: bad w value\n", thisfunc);

  if (handle < 0 || handle > ntimers)
    printf ("%s: bad handle value\n", handle);

  wi = FLATTEN_TIMERS (w, handle);
  timer = timers[wi];

  if ( timer.onflg )
    printf ("%s: onflg was on\n", thisfunc); // Invert logic for better OHD est.

  if (timer.recurselvl > 0) {
    --timer.recurselvl;
    ++timer.count;
  }

  // Last 3 args are timestamp, w, smid
  if (update_stats_gpu (handle, &timer, timer.wall.last, 0, 0) != 0)
    printf ("%s: problem with update_stats_gpu\n", thisfunc);
  timers[wi] = timer;
}

__global__ void GPTLget_memstats_gpu (float *regionmem, float *timernamemem)
{
  *regionmem    = (float) maxwarps * maxtimers * sizeof (Timer);
  *timernamemem = (float)            maxtimers * sizeof (Timername);
  return;
}

__device__ int GPTLmy_sleep (float seconds)
{
  volatile long long start, now;
  volatile double delta;
  static const char *thisfunc = "GPTLmy_sleep";

  if (gpu_hz == 0.)
    return GPTLerror_1s ("%s: need to set gpu_hz via call to GPTLinitialize_gpu() first\n",
			 thisfunc);

  start = clock64();
  do {
    now = clock64();
    delta = (now - start) / gpu_hz;
  } while (delta < seconds);

  // For some reason, w/o syncthreads, ACC tests often sleep much less than 1 sec
  // But CUDA tests all seem to work fine
  // __syncthreads();
  return SUCCESS;
}

__device__ void GPTLdummy_gpu ()
{
  return;
}

#ifdef PRINTNEG
__device__ static void prbits8 (uint64_t val)
{
  uint64_t mask = 1;
  char chars[64];
  
  int i;

  for (i = 0; i < 64; ++i) {
    if ((val & mask) == 0) 
      chars[i] = '0';
    else
      chars[i] = '1';
    val >>= 1;
  }
  
  for (i = 0; i < 64; ++i) {
    printf ("%c", chars[63-i]);
    if ((i+1) % 8 == 0)
      printf (" ");
  }
  printf ("\n");
}
#endif
  
__device__ void GPTLwhoami (const char *caller)
{
  int blockId = blockIdx.x 
    + blockIdx.y * gridDim.x 
    + gridDim.x * gridDim.y * blockIdx.z; 
  int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.z * (blockDim.x * blockDim.y))
    + (threadIdx.y * blockDim.x)
    + threadIdx.x;
  printf ("%s: blockId=%d threadId=%d\n", caller, blockId, threadId);
}
}
