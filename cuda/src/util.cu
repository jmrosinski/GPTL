#include "hip/hip_runtime.h"
/*
** $Id: util.c,v 1.13 2010-01-01 01:34:07 rosinski Exp $
*/

#include "config.h" // Must be first include.
#include "util.h"
#include "init_final.h"
#include "api.h"
#include "output.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>  // LLONG_MAX

__device__ static const int max_errors = 100;     // max number of error print msgs
__device__ static volatile int num_errors = 0;    // number of times error was called
__device__ static volatile int locmutex = 0;      // critical section unscrambles printf output

namespace util {
  __device__ void get_mutex (volatile int *mutex)
  {
    bool isSet;
    do {
      // If mutex is 0, grab by setting = 1
      // If mutex is 1, it stays 1 and isSet will be false
      isSet = atomicCAS ((int *) mutex, 0, 1) == 0;
    } while ( !isSet);   // exit the loop after critical section executed
  }
 
  __device__ void free_mutex (volatile int *mutex)
  {
    *mutex = 0;
  }

  /*
  ** error routines: print a message and return failure
  **
  ** Return value: -1 (failure)
  */
  __device__ int error_1s (const char *fmt, const char *str)
  {
    if (num_errors < max_errors) {
      get_mutex (&locmutex);
      (void) printf ("GPTL error:");
      (void) printf (fmt, str);
      ++num_errors;
      if (num_errors >= max_errors)
      (void) printf ("Truncating further error print now after %d msgs\n", num_errors);
      free_mutex (&locmutex);
    }
    return -1;
  }

  __device__ int error_2s (const char *fmt, const char *str1, const char *str2)
  {
    if (num_errors < max_errors) {
    get_mutex (&locmutex);
    (void) printf ("GPTL error:");
    (void) printf (fmt, str1, str2);
    ++num_errors;
    if (num_errors >= max_errors)
      (void) printf ("Truncating further error print now after %d msgs\n", num_errors);
    free_mutex (&locmutex);
    }
    return -1;
  }

  __device__ int error_1s1d (const char *fmt, const char *str1, const int arg)
  {
    if (num_errors < max_errors) {
      get_mutex (&locmutex);
      (void) printf ("GPTL error:");
      (void) printf (fmt, str1, arg);
      ++num_errors;
      if (num_errors >= max_errors)
	(void) printf ("Truncating further error print now after %d msgs\n", num_errors);
      free_mutex (&locmutex);
    }
    return -1;
  }

  __device__ int error_2s1d (const char *fmt, const char *str1, const char *str2, const int arg1)
  {
    if (num_errors < max_errors) {
      get_mutex (&locmutex);
      (void) printf ("GPTL error:");
      (void) printf (fmt, str1, str2, arg1);
      ++num_errors;
      if (num_errors >= max_errors)
	(void) printf ("Truncating further error print now after %d msgs\n", num_errors);
      free_mutex (&locmutex);
    }
    return -1;
  }

  __device__ int error_2s3d (const char *fmt, const char *str1, const char *str2,
			     const int arg1, const int arg2, const int arg3)
  {
    if (num_errors < max_errors) {
      get_mutex (&locmutex);
      (void) printf ("GPTL error:");
      (void) printf (fmt, str1, str2, arg1, arg2, arg3);
      ++num_errors;
      if (num_errors >= max_errors)
	(void) printf ("Truncating further error print now after %d msgs\n", num_errors);
      free_mutex (&locmutex);
    }
    return -1;
  }

  __device__ int error_1s2d (const char *fmt, const char *str1, const int arg1, const int arg2)
  {
    if (num_errors < max_errors) {
      get_mutex (&locmutex);
      (void) printf ("GPTL error:");
      (void) printf (fmt, str1, arg1, arg2);
      ++num_errors;
      if (num_errors >= max_errors)
	(void) printf ("Truncating further error print now after %d msgs\n", num_errors);
      free_mutex (&locmutex);
    }
    return -1;
  }
  
  // note_gpu: print a note
  __device__ void note_gpu (const char *str)
  {
    (void) printf ("GPTLnote_gpu: %s\n", str);
  }

  // reset_errors: reset error state to no errors
  __device__ void reset_errors_gpu (void)
  {
    num_errors = 0;
  }

  // maxwarpid_timed is needed both on the CPU and on the GPU
  __global__ void get_maxwarpid_timed (int *maxwarpid_timed)
  {
    *maxwarpid_timed = 0;
    // Start w loop from 1 because maxwarpid_timed already inited to 0
    // Start handle loop from 1 since0 is "phantom" timer GPTL_ROOT
    for (int w = 1; w < init_final::maxwarps; ++w) {
      for (int i = 1; i <= api::ntimers; ++i) {
	int wi = FLATTEN_TIMERS (w,i);
	if (api::timers[wi].count > 0 && w >= *maxwarpid_timed)
	  *maxwarpid_timed = w;
      }
    }
  }

  // get_maxwarpid_found is needed to return the result into a hipMallocManaged variable
  __global__ void get_maxwarpid_found (int *maxwarpid_found)
  {
    *maxwarpid_found = api::maxwarpid_found;
  }
  
  /*
  ** reset_gpu: __global__ routine reset a single timer to zero for all warps.
  **   Currently only called by GPTLprint_gpustats
  **
  ** Return argument: *global_retval 0 (success) -1 (failure)
  */
  __global__ void reset_gpu (const int handle, int *global_retval)
  {
    int w, wi;
    static const char *thisfunc = "reset_gpu";
    
    *global_retval = 0;
    if (handle < 0 || handle > api::ntimers) {
      (void) error_1s1d ("%s: bad handle %d\n", thisfunc, handle);
      *global_retval = -1;
      return;
    }

    for (w = 0; w < init_final::maxwarps; ++w) {
      wi = FLATTEN_TIMERS(w,handle);
      api::timers[wi].onflg = false;
      api::timers[wi].count = 0;
      memset (&api::timers[wi].wall, 0, sizeof (api::timers[wi].wall));
      api::timers[wi].wall.min = LLONG_MAX;
    }
    return;
  }
}
