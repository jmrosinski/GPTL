#include "hip/hip_runtime.h"
#include "config.h"  // Must be first include.
#include "device.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {

// Return useful GPU properties. Use arg list for SMcount, cores_per_sm, and cores_per_gpu even 
// though they're globals, because this is a user-callable routine
__host__ int GPTLget_gpu_props (int *khz, int *warpsize, int *devnum, int *SMcount,
				int *cores_per_sm, int *cores_per_gpu)
{
  hipDeviceProp_t prop;
  hipError_t err;
  static const char *thisfunc = "GPTLget_gpu_props";

  if ((err = hipGetDeviceProperties (&prop, 0)) != hipSuccess) {
    printf ("%s: error:%s", thisfunc, hipGetErrorString (err));
    return -1;
  }

  *khz      = prop.clockRate;
  *warpsize = prop.warpSize;
  *SMcount  = prop.multiProcessorCount;

  // Begin code derived from stackoverflow to determine cores_per_sm
  // If hip/hip_runtime_api.h and hip/hip_runtime.h is available (it's not currently in PATH)
  // could call _ConvertSMVer2Cores(prop.major, prop.minor) to get cores_per_sm
  // probably also need hip/hip_runtime.h
  switch (prop.major){
  case 2: // Fermi
    if (prop.minor == 1)
      *cores_per_sm = 48;
    else
      *cores_per_sm = 32;
    break;
  case 3: // Kepler
    *cores_per_sm = 192;
    break;
  case 5: // Maxwell
    *cores_per_sm = 128;
    break;
  case 6: // Pascal
    if ((prop.minor == 1) || (prop.minor == 2))
      *cores_per_sm = 128;
    else if (prop.minor == 0)
      *cores_per_sm = 64;
    else
      printf("Unknown device type\n");
    break;
  case 7: // Volta and Turing
    if ((prop.minor == 0) || (prop.minor == 5))
      *cores_per_sm = 64;
    else
      printf("Unknown device type\n");
    break;
  case 8: // Ampere
    if (prop.minor == 0)
      *cores_per_sm = 64;
    else
      printf("Unknown device type\n");
    break;
  default:
    printf("Unknown device type\n"); 
    break;
  }
  // End code derived from stackoverflow to determine cores_per_sm
  
  // Use _ConvertSMVer2Cores when it is available from nvidia
  //  cores_per_gpu = _ConvertSMVer2Cores (prop.major, prop.minor) * prop.multiProcessorCount);
  *cores_per_gpu = *cores_per_sm * (*SMcount);
  
  err = hipGetDevice (devnum);  // device number
  return 0;
}

__host__ int GPTLcudadevsync (void)
{
  hipDeviceSynchronize ();
  return 0;
}

// The need for these 2 wrapping functions enables gptl.c above to be built with a pure C compiler
// and therefore not require a .cu extension, which itself can cause problems when CUDA is not
// in play.
__host__ int GPTLreset_all_gpu_fromhost (void)
{
  static int *global_retval = 0; // return code from __global__ function

  if (global_retval == 0)  // Unallocated means first call
    // Create space for a "return" value for __global__functions to be checked on CPU
    gpuErrchk (hipMallocManaged (&global_retval, sizeof (int)));

  *global_retval = 0;
  GPTLreset_all_gpu <<<1,1>>> (global_retval);
  hipDeviceSynchronize ();
  if (*global_retval != 0)
    printf ("GPTLreset_all_gpu_fromhost: Failure from GPTLreset_all_gpu\n");
  return *global_retval;
}

__host__ int GPTLfinalize_gpu_fromhost (void)
{
  GPTLfinalize_gpu <<<1,1>>> ();
  return 0;
}
}
