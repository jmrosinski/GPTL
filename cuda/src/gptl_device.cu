#include "hip/hip_runtime.h"
/*
** gptl.cu
** Author: Jim Rosinski
**
** Main file contains most CUDA GPTL functions
*/

#include "config.h" // Must be first include.

#include <stdio.h>
#include <string.h>        // memcpy
#include <stdint.h>        // uint types
#include <hip/hip_runtime.h>

#include "device.h"
#include "gptl_cuda.h"

#define FLATTEN_TIMERS(SUB1,SUB2) (SUB1)*maxtimers + (SUB2)

__device__ static Timer *timers = 0;            // array (also linked list) of timers
__device__ static Timername *timernames;        // array of timer names
__device__ static int max_name_len;             // max length of timer name
__device__ static int ntimers = 0;              // number of timers
__device__ __constant__ static int maxtimers;   // max number of timers
__device__ static int maxwarps = -1;            // max warps
__device__ static int maxwarpid_found = 0;      // number of warps found : init to 0
__device__ static bool initialized = false;     // GPTLinitialize has been called
__device__ static bool verbose = false;         // output verbosity
__device__ static double gpu_hz = 0.;           // clock freq
__device__ int warpsize = 0;                    // warp size
__device__ static volatile int mutex = 0;       // critical section unscrambles printf output

extern "C" {

// Local function prototypes
__global__ static void initialize_gpu (const int, const int, const double, Timer *,
				       Timername *, const int);
__device__ static inline int get_warp_num (void);         // get 0-based 1d warp number
__device__ static inline int update_stats_gpu (const int, Timer *, const long long, const int,
					       const uint);
__device__ static int my_strlen (const char *);
__device__ static char *my_strcpy (char *, const char *);
__device__ static int my_strcmp (const char *, const char *);
__device__ static void start_misc (int, const int);
__device__ static void stop_misc (int w, const int handle);
__device__ static void init_gpustats (Gpustats *, int);
__device__ static void fill_gpustats (Gpustats *, int, int);
// Defining PRINTNEG will print to stdout whenever a negative interval (stop minus start) is
// encountered. Only useful when non-zero negative intervals are reported in timing output
// Should be turned OFF normally--very expensive even when no negatives found.
#undef PRINTNEG
#ifdef PRINTNEG
__device__ static void prbits8 (uint64_t);
#endif

/* VERBOSE is a debugging ifdef local to the rest of this file */
#undef VERBOSE

__host__ int GPTLinitialize_gpu (const int verbose_in,
				 const int maxwarps_in,
				 const int maxtimers_in,
				 const double gpu_hz_in,
				 const int warpsize_in)
{
  size_t nbytes;  // number of bytes to allocate

  // Issue hipMalloc from CPU, and pass address to GPU to avoid mem problems: When run from
  // __global__ routine, mallocable memory is severely decreased for some reason.
  static Timer *timers_cpu = 0;          // array of timers
  static Timername *timernames_cpu = 0; // array of timer names

  // Set constant memory values: First arg is pass by reference so no "&"
  gpuErrchk (hipMemcpyToSymbol(HIP_SYMBOL(maxtimers),   &maxtimers_in,    sizeof (int)));

  nbytes = maxwarps_in * maxtimers_in * sizeof (Timer);
  gpuErrchk (hipMalloc (&timers_cpu, nbytes));

  nbytes =               maxtimers_in * sizeof (Timername);
  gpuErrchk (hipMalloc (&timernames_cpu, nbytes));

  initialize_gpu <<<1,1>>> (verbose_in,
			    maxwarps_in,
			    gpu_hz_in,
			    timers_cpu,
			    timernames_cpu,
			    warpsize_in);
  // This should flush any existing print buffers
  hipDeviceSynchronize ();
  return 0;
}

/*
** GPTLinitialize_gpu (): Initialization routine must be called from single-threaded
**   region before any other timing routines may be called.  The need for this
**   routine could be eliminated if not targetting timing library for threaded
**   capability. 
*/
__global__ static void initialize_gpu (const int verbose_in,
				       const int maxwarps_in,
				       const double gpu_hz_in,
				       Timer *timers_cpu,
				       Timername *timernames_cpu,
				       const int warpsize_in)
{
  int w, wi;        // warp, flattened indices
  long long t1, t2; // returned from underlying timer
  static const char *thisfunc = "initialize_gpu";

#ifdef VERBOSE
  printf ("Entered %s\n", thisfunc);
#endif
  if (initialized) {
    (void) GPTLerror_1s ("%s: has already been called\n", thisfunc);
    return;
  }

  // Set global vars from input args
  verbose    = verbose_in;
  maxwarps   = maxwarps_in;
  gpu_hz     = gpu_hz_in;
  warpsize   = warpsize_in;
  timers     = timers_cpu;
  timernames = timernames_cpu;

  // Initialize timers
  ntimers = 0;
  max_name_len = 0;
  for (w = 0; w < maxwarps; ++w) {
    wi = FLATTEN_TIMERS(w,0);
    memset (&timers[wi], 0, maxtimers * sizeof (Timer));
  }
  // Make a timer "GPTL_ROOT" to ensure no orphans, and to simplify printing.
  memcpy (timernames[0].name, "GPTL_ROOT", 9+1);

  if (verbose) {
    t1 = clock64 ();
    t2 = clock64 ();
    if (t1 > t2)
      printf ("GPTL %s: negative delta-t=%lld\n", thisfunc, t2-t1);

    printf ("Per call overhead est. t2-t1=%g should be near zero\n", (float) (t2-t1));
    printf ("Underlying wallclock timing routine is clock64\n");
  }

  initialized = true;
}

/*
** GPTLfinalize_gpu (): Finalization routine must be called from single-threaded
**   region. Free all malloc'd space
*/
__global__ void GPTLfinalize_gpu (void)
{
  static const char *thisfunc = "GPTLfinalize_gpu";

  if ( ! initialized) {
    (void) GPTLerror_1s ("%s: initialization was not completed\n", thisfunc);
    return;
  }

  free (timers);
  free (timernames);
  
  GPTLreset_errors_gpu ();

  // Reset initial values
  timers = 0;
  timernames = 0;
  max_name_len = 0;
  initialized = false;
  verbose = false;
}

/*
** GPTLinit_handle_gpu: Initialize a handle for further use by GPTLstart_gpu() and GPTLstop_gpu()
**
** Input arguments:
**   name: timer name
**
** Output arguments:
**   handle: Index into array for "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLinit_handle_gpu (const char *name,     // timer name
				    int *handle)          // handle
{
  int numchars;      // length of "name"
  int mywarp;        // my warp number
  int i;
  static const char *thisfunc = "GPTLinit_handle_gpu";

  // Guts of this function are run only by thread 0 of warp 0 due to loop over warps below. 
  // Need to have each timer have the same index for all warps.
  // Nice feature: Can be called by just thread 0 of warp 0, OR NOT
  if ((mywarp = get_warp_num ()) != 0) {
    return SUCCESS;
  }

  // First check if a handle for the requested timer already exists (i=1 skips GPTL_ROOT)
  for (i = 1; i <= ntimers; ++i) {
    if (STRMATCH (name, timernames[i].name)) {
#ifdef DEBUG_PRINT
      printf ("%s name=%s: Returning already existing handle=%d\n", thisfunc, name, i);
#endif
      *handle = i;
      return SUCCESS;
    }
  }
  
  if (ntimers >= maxtimers) {
    return GPTLerror_2s1d ("%s: Too many timers. name=%s maxtimers needs to be incremented from %d\n",
			   thisfunc, name, maxtimers);
  } else {
    numchars = MIN (my_strlen (name), MAX_CHARS);
    max_name_len = MAX (numchars, max_name_len);
    *handle = ++ntimers;
    memcpy (timernames[ntimers].name, name, numchars);
    timernames[ntimers].name[numchars] = '\0';
  }
  //  printf ("%s name=%s: mywarp=%d Returning new handle=%d\n", thisfunc, name, mywarp, *handle);
      
  return SUCCESS;
}

/*
** GPTLstart_gpu: start a timer based on a handle
**
** Input arguments:
**   name: timer name (required when on input, handle=0)
**   handle: pointer to timer matching "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLstart_gpu (const int handle)
{
  Timer *ptr;        // linked list pointer
  int w;             // warp index (of this thread)
  int wi;            // flattened 2d index for warp number and timer name

  static const char *thisfunc = "GPTLstart_gpu";

#ifdef ENABLE_GPUCHECKS
  if ( ! initialized)
    return GPTLerror_1s1d ("%s handle=%d: GPTLinitialize_gpu has not been called\n", 
			   thisfunc, handle);
#endif
  w = get_warp_num ();

  // Return if not thread 0 of the warp, or warpId is outside range of available timers
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    return SUCCESS;

#ifdef VERBOSE
  printf ("Entered %s w=%d handle=%d\n", thisfunc, w, handle);
#endif

  // Input handle should be a positive integer not greater than ntimers (0 accepted for GPTL_ROOT)
#ifdef ENABLE_GPUCHECKS
  if (handle < 0 || handle > ntimers)
    return GPTLerror_1s1d ("%s: Invalid input handle=%d. Perhaps GPTLinit_handle_gpu not called?\n",
			   thisfunc, handle);
#endif
  wi = FLATTEN_TIMERS (w, handle);
  ptr = &timers[wi];
  
  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
#ifdef ENABLE_GPUCHECKS
  if (ptr->onflg) {
    ++ptr->recurselvl;
    return SUCCESS;
  }
#endif

#ifdef DEBUG_PRINT
  printf ("%s: ptr=%p setting onflg=true\n", thisfunc, ptr);
#endif

  // Get the timestamp and smid
  // IMPORTANT: Issue the cmds in sequence because SM changing between clock64() call and getting
  // the SM number would be bad.
  // NOTE: Timing value will be thrown away if SM changes upon stop() call.
  ptr->wall.last = clock64 ();
  asm volatile ("mov.u32 %0, %smid;" : "=r"(ptr->smid));
  ptr->onflg = true;

  return SUCCESS;
}

/*
** GPTLstop_gpu: stop a timer based on a handle
**
** Input arguments:
**   name: timer name (used only for diagnostics)
**   handle: pointer to timer
**
** Return value: 0 (success) or -1 (failure)
*/
__device__ int GPTLstop_gpu (const int handle)
{
  register long long tp1;    // time stamp
  Timer timer;               // local copy of timers[wi]: gives some speedup vs. global array
  int w;                     // warp number for this process
  int wi;                    // flattened (1-d) index into 2-d array [timer][warp]
  uint smid;                 // SM id
  static const char *thisfunc = "GPTLstop_gpu";

#ifdef ENABLE_GPUCHECKS
  if ( ! initialized)
    return GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);
#endif
  
  w = get_warp_num ();

  // Return if not thread 0 of the warp, or warpId is outside range of available timers
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    return SUCCESS;

#ifdef VERBOSE
  printf ("Entered %s w=%d handle=%d\n", thisfunc, w, handle);
#endif

#ifdef ENABLE_GPUCHECKS
  // Input handle should be a positive integer not greater than ntimers (0 accepted for GPTL_ROOT)
  if (handle < 0 || handle > ntimers)
    return GPTLerror_1s1d ("%s: Invalid input handle=%d. Perhaps GPTLinit_handle_gpu not called?\n",
			   thisfunc, handle);
#endif
  // Get the timestamp and smid
  // IMPORTANT: Issue the cmds in sequence because SM changing between clock64() call and getting
  // the SM number would be bad.
  // NOTE: Timing value will be thrown away if SM changed from start() call.
  tp1 = clock64 ();
  asm ("mov.u32 %0, %smid;" : "=r"(smid));

  wi = FLATTEN_TIMERS (w, handle);
  timer = timers[wi];

#ifdef ENABLE_GPUCHECKS
  if ( ! timer.onflg )
    return GPTLerror_2s ("%s: timer %s was already off.\n", thisfunc, timernames[handle].name);
#endif
  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
#ifdef ENABLE_GPUCHECKS
  if (timer.recurselvl > 0) {
    --timer.recurselvl;
    ++timer.count;
    timers[wi] = timer;
    return SUCCESS;
  }
#endif
  if (update_stats_gpu (handle, &timer, tp1, w, smid) != 0)
    return GPTLerror_1s ("%s: error from update_stats_gpu\n", thisfunc);
#ifdef DEBUG_PRINT
  printf ("%s: handle=%d count=%d\n", thisfunc, handle, (int) timer.count);
#endif
  timers[wi] = timer;
  
  return SUCCESS;
}

/*
** update_stats_gpu: update stats inside ptr. Called by GPTLstop_gpu, GPTLstop_handle_gpu
**
** Input arguments:
**   ptr: pointer to timer
**   tp1: input time stamp
**   w: warp index
**
** Return value: 0 (success) or GPTLerror (failure)
*/

__device__ static inline int update_stats_gpu (const int handle,
					       Timer *ptr, 
					       const long long tp1, 
					       const int w,
					       const uint smid)
{
  register long long delta;           // time diff from start()
  static const char *thisfunc = "update_stats_gpu";
#ifdef DEBUG_PRINT
  printf ("%s: ptr=%p setting onflg=false\n", thisfunc, ptr);
#endif

  ptr->onflg = false;
  delta = tp1 - ptr->wall.last;
#ifdef ENABLE_GPUCHECKS
  if (smid != ptr->smid) {
    printf ("GPTL %s: name=%s w=%d sm changed from %d to %d: new kernel? \n"
	    "TIMINGS WITH Bad_SM > 0 PROBABLY INACCURATE.\n"
	    "NEGATIVE STOP MINUS START INCIDENTS WILL BE SKIPPED.\n", 
	    thisfunc, timernames[handle].name, w, ptr->smid, smid);
    ++ptr->badsmid_count;
  }
#endif
  if (delta < 0) {
#ifdef PRINTNEG
    bool isSet; 
    // Use critical section so printf from multiple SMs don't get scrambled
    do {
      // If mutex is 0, grab by setting = 1
      // If mutex is 1, it stays 1 and isSet will be false
      isSet = atomicCAS ((int *) &mutex, 0, 1) == 0; 
      if (isSet) {  // critical section starts here
	printf ("GPTL: %s name=%s w=%d WARNING NEGATIVE DELTA ENCOUNTERED: %lld-%lld=%lld=%g seconds: IGNORING\n", 
		thisfunc, timernames[handle].name, w, tp1, ptr->wall.last, delta, delta / (-gpu_hz));
	printf ("Bit pattern old:");
	prbits8 ((uint64_t) ptr->wall.last);

	printf ("Bit pattern new:");
	prbits8 ((uint64_t) tp1);
	mutex = 0;     // end critical section by releasing the mutex
      }
    } while ( !isSet); // exit the loop after critical section executed
#endif
    
    ++ptr->negdelta_count;
    return SUCCESS;
  }

  ++ptr->count;
  ptr->wall.accum += delta;
  
  if (ptr->count == 1) {
    ptr->wall.max = delta;
    ptr->wall.min = delta;
  } else {
    if (delta > ptr->wall.max)
      ptr->wall.max = delta;
    if (delta < ptr->wall.min)
      ptr->wall.min = delta;
  }
  return SUCCESS;
}

/*
** GPTLreset_gpu: reset all timers to 0
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__global__ void GPTLreset_gpu (void)
{
  int i;
  int w;
  int wi;
  int maxwarpid_timed;
  static const char *thisfunc = "GPTLreset_gpu";

  if ( ! initialized) {
    (void) GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);
    return;
  }

  if (get_warp_num () != 0)
    return;

  maxwarpid_timed = GPTLget_maxwarpid_timed ();

  for (w = 0; w <= maxwarpid_timed; ++w) {
    for (i = 0; i < maxtimers; ++i) {
      wi = FLATTEN_TIMERS(w,i);
      timers[wi].onflg = false;
      timers[wi].count = 0;
      memset (&timers[wi].wall, 0, sizeof (timers[wi].wall));
    }
  }

  if (verbose)
    printf ("%s: accumulators for all GPU timers set to zero\n", thisfunc);
}

__device__ static inline int get_warp_num ()
{
  int threadId;
  int warpId;

  threadId = threadIdx.x
        +  blockDim.x  * threadIdx.y
        +  blockDim.x  *  blockDim.y  * threadIdx.z
        +  blockDim.x  *  blockDim.y  *  blockDim.z  * blockIdx.x
        +  blockDim.x  *  blockDim.y  *  blockDim.z  *  gridDim.x  * blockIdx.y
        +  blockDim.x  *  blockDim.y  *  blockDim.z  *  gridDim.x  *  gridDim.y  * blockIdx.z;

  // Only thread 0 of the warp will be timed
  if (threadId % warpsize != 0)
    return NOT_ROOT_OF_WARP;

  warpId = threadId / warpsize;

  // Setting maxwarpid_found is a race condition that is ignored due to efficiency considerations
  // It is only printed as an estimate when GPTLpr is called.
#ifdef ENABLE_FOUND
  if (warpId+1 > maxwarpid_found)
    maxwarpid_found = warpId;
#endif
  
  if (warpId > maxwarps-1)
    return WARPID_GT_MAXWARPS;

  return warpId;
}

__device__ int GPTLget_maxwarpid_timed (void)
{
  int wi;
  int maxwarpid_timed = 0;
  static const char *thisfunc = "GPTLget_maxwarpid_timed";

  if (get_warp_num () != 0) {
    (void) GPTLerror_1s ("%s: must only be called by thread 0 of warp 0\n", thisfunc);
    return -1;
  }

  for (int w = 0; w < maxwarps; ++w) {
    for (int i = ntimers; i > 0; --i) {
      wi = FLATTEN_TIMERS(w,i);
      if (timers[wi].count > 0 && w > maxwarpid_timed)
	maxwarpid_timed = w;
    }
  }
  return maxwarpid_timed;
}

__device__ int GPTLget_wallclock_gpu (const int handle,
				      double *accum, double *max, double *min)
{
  int w;
  int wi;
  static const char *thisfunc = "GPTLget_wallclock_gpu";
  
  if ( ! initialized)
    (void) GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);

  if (gpu_hz == 0.)
    (void) GPTLerror_1s ("%s: gpu_hz has not been set\n", thisfunc);

  w = get_warp_num ();
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    return SUCCESS;

  if (handle < 0 || handle > ntimers)
    return GPTLerror_1s1d ("%s: bad handle=%d\n", thisfunc, handle);

  wi = FLATTEN_TIMERS (w, handle);
  
  *accum = timers[wi].wall.accum / gpu_hz;
  *max   = timers[wi].wall.max   / gpu_hz;
  *min   = timers[wi].wall.min   / gpu_hz;
  return 0;
}

//JR want to use variables to dimension arrays but nvcc is not C99 compliant
__global__ void GPTLfill_gpustats (Gpustats *gpustats, 
				   int *max_name_len_out,
				   int *ngputimers)
{
  int w;
  int n;
  int maxwarpid_timed;
  static const char *thisfunc = "GPTLfill_gpustats";

  if ( ! initialized) {
    (void) GPTLerror_1s ("%s: GPTLinitialize_gpu has not been called\n", thisfunc);
    return;
  }

  if (get_warp_num () != 0) {
    (void) GPTLerror_1s ("%s: must only be called by thread 0 of warp 0\n", thisfunc);
    return;
  }

  maxwarpid_timed = GPTLget_maxwarpid_timed ();
  *max_name_len_out = max_name_len;
  *ngputimers = ntimers;

  // Step 1: process entries for all warps based on those in warp 0
  // gpustats starts at 0. timers start at 1
  for (n = 0; n < ntimers; ++n) {
    init_gpustats (&gpustats[n], n+1);
    for (w = 1; w <= maxwarpid_timed; ++w) {
      fill_gpustats (&gpustats[n], n+1, w);
    }
  }

#ifdef DEBUG_PRINT
  printf ("%s: ngputimers=%d\n", thisfunc, n);
  for (n = 0; n < *ngputimers; ++n) {
    printf ("%s: timer=%s accum_max=%lld accum_min=%lld count_max=%d nwarps=%d\n", 
	    thisfunc, gpustats[n].name, gpustats[n].accum_max, gpustats[n].accum_min, gpustats[n].count_max, gpustats[n].nwarps);
  }
#endif
  return;
}

__device__ static void init_gpustats (Gpustats *gpustats, int idx)
{
  const int w = 0;
  (void) my_strcpy (gpustats->name, timernames[idx].name);
  gpustats->count  = timers[idx].count;
  if (timers[idx].count > 0)
    gpustats->nwarps = 1;
  else
    gpustats->nwarps = 0;

  gpustats->accum_max      = timers[idx].wall.accum;
  gpustats->accum_max_warp = w;

  gpustats->accum_min      = timers[idx].wall.accum;
  gpustats->accum_min_warp = w;

  gpustats->count_max      = timers[idx].count;
  gpustats->count_max_warp = w;

  gpustats->count_min      = timers[idx].count;
  gpustats->count_min_warp = w;

  gpustats->negdelta_count_max       = timers[idx].negdelta_count;
  gpustats->negdelta_count_max_warp  = w;
  gpustats->negdelta_nwarps          = timers[idx].negdelta_count  > 0 ? 1 : 0;

  gpustats->badsmid_count  = timers[idx].badsmid_count;
}

__device__ static void fill_gpustats (Gpustats *gpustats, int idx, int w)
{
  int wi = FLATTEN_TIMERS (w,idx);
  
  if (timers[wi].count > 0) {
    gpustats->count += timers[wi].count;
    ++gpustats->nwarps;

    if (timers[wi].wall.accum > gpustats->accum_max) {
      gpustats->accum_max      = timers[wi].wall.accum;
      gpustats->accum_max_warp = w;
    }
    
    if (timers[wi].wall.accum < gpustats->accum_min) {
      gpustats->accum_min      = timers[wi].wall.accum;
      gpustats->accum_min_warp = w;
    }
    
    if (timers[wi].count > gpustats->count_max) {
      gpustats->count_max      = timers[wi].count;
      gpustats->count_max_warp = w;
    }
    
    if (timers[wi].count < gpustats->count_min) {
      gpustats->count_min      = timers[wi].count;
      gpustats->count_min_warp = w;
    }
    
    if (timers[wi].negdelta_count > gpustats->negdelta_count_max) {
      gpustats->negdelta_count_max      = timers[wi].negdelta_count;
      gpustats->negdelta_count_max_warp = w;
    }

    if (timers[wi].negdelta_count > 0)
      ++gpustats->negdelta_nwarps;

    gpustats->badsmid_count += timers[wi].badsmid_count;
  }
}

__device__ static int my_strlen (const char *str)
{
  const char *s;
  for (s = str; *s; ++s);
  return(s - str);
}

__device__ static inline char *my_strcpy (char *dest, const char *src)
{
  char *ret = dest;

  while (*src != '\0')
    *dest++ = *src++;
  *dest = '\0';
  return ret;
}

//JR Both of these have about the same performance
__device__ static int my_strcmp (const char *str1, const char *str2)
{
#ifndef MINE
  while (*str1 == *str2) {
    if (*str1 == '\0')
      break;
    ++str1;
    ++str2;
  }
  return (int) (*str1 - *str2);
#else
  register const unsigned char *s1 = (const unsigned char *) str1;
  register const unsigned char *s2 = (const unsigned char *) str2;
  register unsigned char c1, c2;
 
  do {
      c1 = (unsigned char) *s1++;
      c2 = (unsigned char) *s2++;
      if (c1 == '\0')
	return c1 - c2;
  } while (c1 == c2); 
  return c1 - c2;
#endif
}

// Overhead estimate functions start here
/*
** GPTLget_overhead: return current status info about a timer. If certain stats are not enabled, 
** they should just have zeros in them.
** 
** Output args:
**   get_warp_num_ohd: Getting my warp index
**   utr_ohd:            Underlying timer routine
**   self_ohd:           Estimate of GPTL-induced overhead in the timer itself (included in "Wallclock")
**   parent_ohd:         Estimate of GPTL-induced overhead for the timer which appears in its parents
*/
__global__ void GPTLget_overhead_gpu (int *maxwarpid_timed_out,
				      int *maxwarpid_found_out,
				      long long *get_warp_num_ohd,  // Getting my warp index
				      long long *startstop_ohd,     // start/stop pair
				      long long *utr_ohd,           // Underlying timing routine
				      long long *start_misc_ohd,    // misc start code
				      long long *stop_misc_ohd,     // misc stop code
				      long long *self_ohd,          // OHD in timer itself
				      long long *parent_ohd,        // OHD in parent
				      long long *my_strlen_ohd,
				      long long *STRMATCH_ohd)
{
  volatile uint smid;         // SM id
  long long t1, t2;           // Initial, final timer values
  int i;
  int ret;
  int mywarp;                 // our warp number
  char name[MAX_CHARS+1];     // Name to be used for various OHD tests
  char samename[MAX_CHARS+1]; // Copy of "name" for STRMATCH test

  *maxwarpid_timed_out = GPTLget_maxwarpid_timed ();
  *maxwarpid_found_out = maxwarpid_found;
  
  // Define name to be used in OHD estimates. Use GPTL_ROOT because it's always there
  my_strcpy (name, timernames[0].name); // GPTL_ROOT
  my_strcpy (samename, name);

  /*
  ** Gather timings by running each test 1000 times
  ** First: start/stop overhead 
  */
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    ret = GPTLstart_gpu (0);
    ret = GPTLstop_gpu (0);
  }
  t2 = clock64();
  startstop_ohd[0] = (t2 - t1) / 1000;

  // get_warp_num overhead. Need a bogus computation or compiler may optimize out the code
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    if ((mywarp = get_warp_num ()) < -999)
      get_warp_num_ohd[0] = -999;
  }
  t2 = clock64();
  get_warp_num_ohd[0] = (t2 - t1) / 1000;

  // utr overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    asm volatile ("mov.u32 %0, %smid;" : "=r"(smid));
    t2 = clock64();
  }
  *utr_ohd = (t2 - t1) / 1000;

  // start misc overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    start_misc (0, 0);  // w, handle (handle=0 is GPTL_ROOT)
  }
  t2 = clock64();
  start_misc_ohd[0] = (t2 - t1) / 1000;

  // stop misc overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    stop_misc (0, 0);  // w, handle (handle=0 is GPTL_ROOT)
  }
  t2 = clock64();
  stop_misc_ohd[0] = (t2 - t1) / 1000;

  // Self and parent OHD estimates: A few settings at the end of GPTLstart_gpu should instead be 
  // applied to parent. A few settings at the beginning of GPTLstop_gpu should instead be
  // applied to self. But those errors are likely minor.
  self_ohd[0]   = utr_ohd[0] + start_misc_ohd[0];
  parent_ohd[0] = utr_ohd[0] + 2*get_warp_num_ohd[0] + stop_misc_ohd[0];

  // my_strlen overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    ret = my_strlen (name);
  }
  t2 = clock64();
  *my_strlen_ohd = (t2 - t1) / 1000;

  // STRMATCH overhead
  t1 = clock64();
  for (i = 0; i < 1000; ++i) {
    ret = STRMATCH (samename, name);
  }
  t2 = clock64();
  *STRMATCH_ohd = (t2 - t1) / 1000;
  return;
}

__device__ static void start_misc (int w, const int handle)
{
  int wi;
  Timer *ptr;
  static const char *thisfunc = "startmisc";

#ifdef ENABLE_GPUCHECKS
  if ( ! initialized)
    printf ("%s: ! initialized\n", thisfunc);
#endif
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    printf ("%s: bad w value\n", thisfunc);

#ifdef ENABLE_GPUCHECKS
  if (handle < 0 || handle > ntimers)
    printf ("%s: bad handle value %d\n", thisfunc, handle);
#endif
  wi = FLATTEN_TIMERS (w, handle);
  ptr = &timers[wi];

#ifdef ENABLE_GPUCHECKS
  if (ptr->onflg) {
    ++ptr->recurselvl;
    printf ("%s: onflg should be off\n", thisfunc);
    ptr->smid = 0;
    ptr->wall.last = 0L;
  }
#endif
  ptr->onflg = false;  // GPTLstart actually sets this true but set false for better OHD est.
}

__device__ static void stop_misc (int w, const int handle)
{
  int wi;
  Timer timer;
  static const char *thisfunc = "stopmisc";

#ifdef ENABLE_GPUCHECKS
  if ( ! initialized)
    printf ("%s: ! initialized\n", thisfunc);
  if (w == NOT_ROOT_OF_WARP || w == WARPID_GT_MAXWARPS)
    printf ("%s: bad w value\n", thisfunc);

  if (handle < 0 || handle > ntimers)
    printf ("%s: bad handle value %d\n", thisfunc, handle);
#endif

  wi = FLATTEN_TIMERS (w, handle);
  timer = timers[wi];

#ifdef ENABLE_GPUCHECKS
  if ( timer.onflg )
    printf ("%s: onflg was on\n", thisfunc); // Invert logic for better OHD est.
  if (timer.recurselvl > 0) {
    --timer.recurselvl;
    ++timer.count;
  }
#endif

  // Last 3 args are timestamp, w, smid
  if (update_stats_gpu (handle, &timer, timer.wall.last, 0, 0) != 0)
    printf ("%s: problem with update_stats_gpu\n", thisfunc);
  timers[wi] = timer;
}

__global__ void GPTLget_memstats_gpu (float *regionmem, float *timernamemem)
{
  *regionmem    = (float) maxwarps * maxtimers * sizeof (Timer);
  *timernamemem = (float)            maxtimers * sizeof (Timername);
  return;
}

__device__ int GPTLmy_sleep (float seconds)
{
  volatile long long start, now;
  volatile double delta;
  static const char *thisfunc = "GPTLmy_sleep";

  int mywarp = get_warp_num ();

  // Only sleep if we're root of warp
  if (mywarp == NOT_ROOT_OF_WARP)
    return SUCCESS;

  if (gpu_hz == 0.)
    return GPTLerror_1s ("%s: need to set gpu_hz via call to GPTLinitialize_gpu() first\n",
			 thisfunc);

  start = clock64();
  do {
    now = clock64();
    delta = (now - start) / gpu_hz;
  } while (delta < seconds);

  return SUCCESS;
}

__device__ void GPTLdummy_gpu ()
{
  return;
}

#ifdef PRINTNEG
__device__ static void prbits8 (uint64_t val)
{
  uint64_t mask = 1;
  char chars[64];
  
  int i;

  for (i = 0; i < 64; ++i) {
    if ((val & mask) == 0) 
      chars[i] = '0';
    else
      chars[i] = '1';
    val >>= 1;
  }
  
  for (i = 0; i < 64; ++i) {
    printf ("%c", chars[63-i]);
    if ((i+1) % 8 == 0)
      printf (" ");
  }
  printf ("\n");
}
#endif
  
__device__ int GPTLget_warp_thread (int *warp, int *thread)
{
  static const char *thisfunc = "GPTLget_warp_thread";
  if ( ! initialized) {
    (void) GPTLerror_1s ("%s: initialization was not completed\n", thisfunc);
    return -1;
  }

  *thread = threadIdx.x
        +  blockDim.x  * threadIdx.y
        +  blockDim.x  *  blockDim.y  * threadIdx.z
        +  blockDim.x  *  blockDim.y  *  blockDim.z  * blockIdx.x
        +  blockDim.x  *  blockDim.y  *  blockDim.z  *  gridDim.x  * blockIdx.y
        +  blockDim.x  *  blockDim.y  *  blockDim.z  *  gridDim.x  *  gridDim.y  * blockIdx.z;
  *warp = (*thread) / warpsize;
  return 0;
}

__device__ int GPTLsliced_up_how (const char *txt)
{
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 &&
      blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
    printf ("GPTLsliced_up_how: %s\n", txt);
    if (blockDim.x > 1)
      printf ("blockDim.x=%d ", blockDim.x);
    if (blockDim.y > 1)
      printf ("blockDim.y=%d ", blockDim.y);
    if (blockDim.z > 1)
      printf ("blockDim.z=%d ", blockDim.z);
    printf ("\n");

    if (gridDim.x > 1)
      printf ("gridDim.x=%d ", gridDim.x);
    if (gridDim.y > 1)
      printf ("gridDim.y=%d ", gridDim.y);
    printf ("\n");
  }
  return 0;
}
}
