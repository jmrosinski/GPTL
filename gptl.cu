#include "hip/hip_runtime.h"
//#define _GLIBCXX_CMATH
/*
** gptl.c
** Author: Jim Rosinski
**
** Main file contains most user-accessible GPTL functions
*/

#ifdef HAVE_MPI
#include <mpi.h>
#endif

#include <stdlib.h>        /* malloc */
#include <sys/time.h>      /* gettimeofday */
#include <sys/times.h>     /* times */
#include <unistd.h>        /* gettimeofday, syscall */
#include <stdio.h>
#include <string.h>        /* memset, strcmp (via STRMATCH) */
#include <ctype.h>         /* isdigit */
#include <hip/hip_runtime.h>

#ifdef HAVE_LIBRT
#include <time.h>
#endif

#ifdef _AIX
#include <sys/systemcfg.h>
#endif

#ifdef HAVE_BACKTRACE
#include <execinfo.h>
#endif

#include "private.h"
#include "gptl.h"

#include <hip/hip_runtime_api.h>
int GPTLcores_per_sm = -1;
int GPTLcores_per_gpu = -1;

static Timer **timers = 0;             /* linked list of timers */
static Timer **last = 0;               /* last element in list */
static int *max_depth;                 /* maximum indentation level encountered */
static int *max_name_len;              /* max length of timer name */
static volatile int nthreads = -1;     /* num threads. Init to bad value */
static volatile int maxthreads = -1;   /* max threads */
static int depthlimit = 99999;         /* max depth for timers (99999 is effectively infinite) */
static volatile bool disabled = false; /* Timers disabled? */
static volatile bool initialized = false;        /* GPTLinitialize has been called */
static volatile bool pr_has_been_called = false; /* GPTLpr_file has been called */
static bool verbose = false;           /* output verbosity */
static bool percent = false;           /* print wallclock also as percent of 1st timers[0] */
static bool dopr_preamble = true;      /* whether to print preamble info */
static bool dopr_threadsort = true;    /* whether to print sorted thread stats */
static bool dopr_multparent = true;    /* whether to print multiple parent info */
static bool dopr_collision = true;     /* whether to print hash collision info */
static bool dopr_memusage = false;     /* whether to include memusage print when auto-profiling */
static int SMcount = -1;               // SM count for each GPU
static int khz = -1;
static int warpsize = -1;

static time_t ref_gettimeofday = -1;   /* ref start point for gettimeofday */
static time_t ref_clock_gettime = -1;  /* ref start point for clock_gettime */
#ifdef _AIX
static time_t ref_read_real_time = -1; /* ref start point for read_real_time */
#endif

#if ( defined THREADED_OMP )

#include <omp.h>
volatile int *GPTLthreadid_omp = 0; /* array of thread ids */

#else

/* Unthreaded case */
int GPTLthreadid = -1;

#endif

typedef struct {
  const Option option;  /* wall, cpu, etc. */
  const char *str;      /* descriptive string for printing */
  bool enabled;         /* flag */
} Settings;

/* Options, print strings, and default enable flags */
static Settings cpustats =      {GPTLcpu,      "Usr       sys       usr+sys   ", false};
static Settings wallstats =     {GPTLwall,     "Wallclock max       min       ", true };
static Settings overheadstats = {GPTLoverhead, "self_OH  parent_OH "           , true };

static Hashentry **hashtable;    /* table of entries */
static long ticks_per_sec;       /* clock ticks per second */
static Timer ***callstack;       /* call stack */
static Nofalse *stackidx;        /* index into callstack: */

static Method method = GPTLfull_tree;  /* default parent/child printing mechanism */

#ifdef HAVE_NANOTIME
static float cpumhz = -1.;                        /* init to bad value */
static double cyc2sec = -1;                       /* init to bad value */
extern "C" {
  static inline long long nanotime (void);          /* read counter (assembler) */
  static float get_clockfreq (void);                /* cycles/sec */
}
static char *clock_source = "Unknown";            /* where clock found */
#endif

#define DEFAULT_TABLE_SIZE 1023
static int tablesize = DEFAULT_TABLE_SIZE;  /* per-thread size of hash table (settable parameter) */
static int tablesizem1 = DEFAULT_TABLE_SIZE - 1;

static double gpu_hz = 0.;       // GPU frequency in cycles per second
static int maxwarps_gpu = DEFAULT_MAXWARPS_GPU;
static int maxtimers_gpu = DEFAULT_MAXTIMERS_GPU;
static int devnum = -1;

#define MSGSIZ 256                          /* max size of msg printed when dopr_memusage=true */
static int rssmax = 0;                      /* max rss of the process */
static bool imperfect_nest;                 /* e.g. start(A),start(B),stop(A) */

/* VERBOSE is a debugging ifdef local to the rest of this file */
#undef VERBOSE

extern "C" {
/* Local function prototypes */
__host__ static void print_titles (int, FILE *);
__host__ static void printstats (const Timer *, FILE *, int, int, bool, double, double);
__host__ static void add (Timer *, const Timer *);
__host__ static void print_multparentinfo (FILE *, Timer *);
__host__ static inline int get_cpustamp (long *, long *);
__host__ static int newchild (Timer *, Timer *);
__host__ static int get_max_depth (const Timer *, const int);
__host__ static int is_descendant (const Timer *, const Timer *);
__host__ static int is_onlist (const Timer *, const Timer *);
__host__ static const char *methodstr (Method);

/* Prototypes from previously separate file threadutil.c */
__host__ static int threadinit (void);                    /* initialize threading environment */
__host__ static void threadfinalize (void);               /* finalize threading environment */
__host__ static inline int get_thread_num (void);         /* get 0-based thread number */

/* These are the (possibly) supported underlying wallclock timers */
__host__ static inline double utr_nanotime (void);
__host__ static inline double utr_mpiwtime (void);
__host__ static inline double utr_clock_gettime (void);
__host__ static inline double utr_read_real_time (void);
__host__ static inline double utr_gettimeofday (void);
__host__ static inline double utr_placebo (void);

__host__ static int init_nanotime (void);
__host__ static int init_mpiwtime (void);
__host__ static int init_clock_gettime (void);
__host__ static int init_read_real_time (void);
__host__ static int init_gettimeofday (void);
__host__ static int init_placebo (void);

__host__ static inline unsigned int genhashidx (const char *);
__host__ static inline Timer *getentry_instr (const Hashentry *, void *, unsigned int *);
__host__ static inline Timer *getentry (const Hashentry *, const char *, unsigned int);
__host__ static void printself_andchildren (const Timer *, FILE *, int, int, double, double);
__host__ static inline int update_parent_info (Timer *, Timer **, int);
__host__ static inline int update_stats (Timer *, const double, const long, const long, const int);
__host__ static int update_ll_hash (Timer *, int, unsigned int);
__host__ static inline int update_ptr (Timer *, const int);
__host__ static int construct_tree (Timer *, Method);

typedef struct {
  const Funcoption option;
  double (*func)(void);
  int (*funcinit)(void);
  const char *name;
} Funcentry;

static Funcentry funclist[] = {
  {GPTLgettimeofday,   utr_gettimeofday,   init_gettimeofday,  "gettimeofday"},
  {GPTLnanotime,       utr_nanotime,       init_nanotime,      "nanotime"},
  {GPTLmpiwtime,       utr_mpiwtime,       init_mpiwtime,      "MPI_Wtime"},
  {GPTLclockgettime,   utr_clock_gettime,  init_clock_gettime, "clock_gettime"},
  {GPTLread_real_time, utr_read_real_time, init_read_real_time,"read_real_time"},     /* AIX only */
  {GPTLplacebo,        utr_placebo,        init_placebo,       "placebo"}      /* does nothing */
};
static const int nfuncentries = sizeof (funclist) / sizeof (Funcentry);
static double (*ptr2wtimefunc)() = 0; /* init to invalid */
static int funcidx = 0;               /* default timer is gettimeofday */

/*
** GPTLsetoption: set option value to true or false.
**
** Input arguments:
**   option: option to be set
**   val:    value to which option should be set (nonzero=true, zero=false)
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLsetoption (const int option,  /* option */
			    const int val)     /* value */
{
  static const char *thisfunc = "GPTLsetoption";

  if (initialized)
    return GPTLerror ("%s: must be called BEFORE GPTLinitialize\n", thisfunc);

  if (option == GPTLabort_on_error) {
    GPTLset_abort_on_error ((bool) val);
    if (verbose)
      printf ("%s: boolean abort_on_error = %d\n", thisfunc, val);
    return 0;
  }

  switch (option) {
  case GPTLcpu:
#ifdef HAVE_TIMES
    cpustats.enabled = (bool) val; 
    if (verbose)
      printf ("%s: cpustats = %d\n", thisfunc, val);
#else
    if (val)
      return GPTLerror ("%s: times() not available\n", thisfunc);
#endif
    return 0;
  case GPTLwall:     
    wallstats.enabled = (bool) val; 
    if (verbose)
      printf ("%s: boolean wallstats = %d\n", thisfunc, val);
    return 0;
  case GPTLoverhead: 
    overheadstats.enabled = (bool) val; 
    if (verbose)
      printf ("%s: boolean overheadstats = %d\n", thisfunc, val);
    return 0;
  case GPTLdepthlimit: 
    depthlimit = val; 
    if (verbose)
      printf ("%s: depthlimit = %d\n", thisfunc, val);
    return 0;
  case GPTLverbose: 
    verbose = (bool) val; 
    if (verbose)
      printf ("%s: boolean verbose = %d\n", thisfunc, val);
    return 0;
  case GPTLpercent: 
    percent = (bool) val; 
    if (verbose)
      printf ("%s: boolean percent = %d\n", thisfunc, val);
    return 0;
  case GPTLdopr_preamble: 
    dopr_preamble = (bool) val; 
    if (verbose)
      printf ("%s: boolean dopr_preamble = %d\n", thisfunc, val);
    return 0;
  case GPTLdopr_threadsort: 
    dopr_threadsort = (bool) val; 
    if (verbose)
      printf ("%s: boolean dopr_threadsort = %d\n", thisfunc, val);
    return 0;
  case GPTLdopr_multparent: 
    dopr_multparent = (bool) val; 
    if (verbose)
      printf ("%s: boolean dopr_multparent = %d\n", thisfunc, val);
    return 0;
  case GPTLdopr_collision: 
    dopr_collision = (bool) val; 
    if (verbose)
      printf ("%s: boolean dopr_collision = %d\n", thisfunc, val);
    return 0;
  case GPTLdopr_memusage: 
    dopr_memusage = (bool) val; 
    if (verbose)
      printf ("%s: boolean dopr_memusage = %d\n", thisfunc, val);
    return 0;
  case GPTLprint_method:
    method = (Method) val; 
    if (verbose)
      printf ("%s: print_method = %s\n", thisfunc, methodstr (method));
    return 0;
  case GPTLsync_mpi:
    if (verbose)
      printf ("%s: boolean sync_mpi = %d\n", thisfunc, val);
    return 0;
  case GPTLmaxthreads:
    if (val < 1)
      return GPTLerror ("%s: maxthreads must be positive. %d is invalid\n", thisfunc, val);
    maxthreads = val;
    return 0;
  case GPTLtablesize:
    if (val < 1)
      return GPTLerror ("%s: tablesize must be positive. %d is invalid\n", thisfunc, val);
    tablesize = val;
    tablesizem1 = val - 1;
    if (verbose)
      printf ("%s: tablesize = %d\n", thisfunc, tablesize);
    return 0;
  case GPTLmaxwarps_gpu:
    if (val < 1)
      return GPTLerror ("%s: maxwarps_gpu must be positive. %d is invalid\n", thisfunc, val);
    maxwarps_gpu = val;
    printf ("%s: maxwarps_gpu = %d\n", thisfunc, maxwarps_gpu);
    return 0;
  case GPTLmaxtimers_gpu:
    if (val < 1)
      return GPTLerror ("%s: maxtimers_gpu must be positive. %d is invalid\n", thisfunc, val);
    maxtimers_gpu = val;
    printf ("%s: maxtimers_gpu = %d\n", thisfunc, maxtimers_gpu);
    return 0;
  default:
    break;
  }

  return GPTLerror ("%s: failure to enable option %d\n", thisfunc, option);
}

/*
** GPTLsetutr: set underlying timing routine.
**
** Input arguments:
**   option: index which sets function
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLsetutr (const int option)
{
  int i;  /* index over number of underlying timer  */
  static const char *thisfunc = "GPTLsetutr";

  if (initialized)
    return GPTLerror ("%s: must be called BEFORE GPTLinitialize\n", thisfunc);

  for (i = 0; i < nfuncentries; i++) {
    if (option == (int) funclist[i].option) {
      if (verbose)
        printf ("%s: underlying wallclock timer = %s\n", thisfunc, funclist[i].name);
      funcidx = i;

      /*
      ** Return an error condition if the function is not available.
      ** OK for the user code to ignore: GPTLinitialize() will reset to gettimeofday
      */

      if ((*funclist[i].funcinit)() < 0)
        return GPTLerror ("%s: utr=%s not available or doesn't work\n", thisfunc, funclist[i].name);
      else
        return 0;
    }
  }
  return GPTLerror ("%s: unknown option %d\n", thisfunc, option);
}

/*
** GPTLinitialize (): Initialization routine must be called from single-threaded
**   region before any other timing routines may be called.  The need for this
**   routine could be eliminated if not targetting timing library for threaded
**   capability. 
**
** return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLinitialize (void)
{
  int i;          /* loop index */
  int t;          /* thread index */
  int ret;        /* return value */
  double t1, t2;  /* returned from underlying timer */
  static const char *thisfunc = "GPTLinitialize";

  if (initialized)
    return GPTLerror ("%s: has already been called\n", thisfunc);

  if (threadinit () < 0)
    return GPTLerror ("%s: bad return from threadinit\n", thisfunc);

  if ((ticks_per_sec = sysconf (_SC_CLK_TCK)) == -1)
    return GPTLerror ("%s: failure from sysconf (_SC_CLK_TCK)\n", thisfunc);

  /* Allocate space for global arrays */
  callstack     = (Timer ***)    GPTLallocate (maxthreads * sizeof (Timer **), thisfunc);
  stackidx      = (Nofalse *)    GPTLallocate (maxthreads * sizeof (Nofalse), thisfunc);
  timers        = (Timer **)     GPTLallocate (maxthreads * sizeof (Timer *), thisfunc);
  last          = (Timer **)     GPTLallocate (maxthreads * sizeof (Timer *), thisfunc);
  max_depth     = (int *)        GPTLallocate (maxthreads * sizeof (int), thisfunc);
  max_name_len  = (int *)        GPTLallocate (maxthreads * sizeof (int), thisfunc);
  hashtable     = (Hashentry **) GPTLallocate (maxthreads * sizeof (Hashentry *), thisfunc);

  /* Initialize array values */
  for (t = 0; t < maxthreads; t++) {
    max_depth[t]    = -1;
    max_name_len[t] = 0;
    callstack[t] = (Timer **) GPTLallocate (MAX_STACK * sizeof (Timer *), thisfunc);
    hashtable[t] = (Hashentry *) GPTLallocate (tablesize * sizeof (Hashentry), thisfunc);
    for (i = 0; i < tablesize; i++) {
      hashtable[t][i].nument = 0;
      hashtable[t][i].entries = 0;
    }

    /* Make a timer "GPTL_ROOT" to ensure no orphans, and to simplify printing. */
    timers[t] = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (timers[t], 0, sizeof (Timer));
    strcpy (timers[t]->name, "GPTL_ROOT");
    timers[t]->onflg = true;
    last[t] = timers[t];

    stackidx[t].val = 0;
    callstack[t][0] = timers[t];
    for (i = 1; i < MAX_STACK; i++)
      callstack[t][i] = 0;
  }

  /* Call init routine for underlying timing routine. */
  if ((*funclist[funcidx].funcinit)() < 0) {
    fprintf (stderr, "%s: Failure initializing %s. Reverting underlying timer to %s\n", 
             thisfunc, funclist[funcidx].name, funclist[0].name);
    funcidx = 0;
  }

  ptr2wtimefunc = funclist[funcidx].func;

  if (verbose) {
    t1 = (*ptr2wtimefunc) ();
    t2 = (*ptr2wtimefunc) ();
    if (t1 > t2)
      fprintf (stderr, "%s: negative delta-t=%g\n", thisfunc, t2-t1);
    printf ("Per call overhead est. t2-t1=%g should be near zero\n", t2-t1);
    printf ("Underlying wallclock timing routine is %s\n", funclist[funcidx].name);
  }

  ret = GPTLget_gpu_props (&khz, &warpsize, &devnum, &SMcount, &GPTLcores_per_sm, &GPTLcores_per_gpu);
  if (warpsize != WARPSIZE)
    return GPTLerror ("%s: warpsize=%d WARPSIZE=%d\n", thisfunc, warpsize, WARPSIZE);
  printf ("%s: device number=%d\n", thisfunc, devnum);

  gpu_hz = khz * 1000.;
  printf ("%s: GPU khz=%d\n", thisfunc, khz);
  ret = GPTLinitialize_gpu (verbose, maxwarps_gpu, maxtimers_gpu, gpu_hz);
  printf ("%s: Returned from GPTLinitialize_gpu\n", thisfunc);
  imperfect_nest = false;
  initialized = true;
  return 0;
}

/*
** GPTLfinalize (): Finalization routine must be called from single-threaded
**   region. Free all malloc'd space
**
** return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLfinalize (void)
{
  int t;                /* thread index */
  int n;                /* array index */
  Timer *ptr, *ptrnext; /* ll indices */
  static const char *thisfunc = "GPTLfinalize";

  if ( ! initialized)
    return GPTLerror ("%s: initialization was not completed\n", thisfunc);

  for (t = 0; t < maxthreads; ++t) {
    for (n = 0; n < tablesize; ++n) {
      if (hashtable[t][n].nument > 0)
        free (hashtable[t][n].entries);
    }
    free (hashtable[t]);
    hashtable[t] = NULL;
    free (callstack[t]);
    for (ptr = timers[t]; ptr; ptr = ptrnext) {
      ptrnext = ptr->next;
      if (ptr->nparent > 0) {
        free (ptr->parent);
        free (ptr->parent_count);
      }
      if (ptr->nchildren > 0)
        free (ptr->children);
      free (ptr);
    }
  }

  free (callstack);
  free (stackidx);
  free (timers);
  free (last);
  free (max_depth);
  free (max_name_len);
  free (hashtable);

  threadfinalize ();
  GPTLreset_errors ();

  /* Reset initial values */
  timers = 0;
  last = 0;
  max_depth = 0;
  max_name_len = 0;
  nthreads = -1;
  maxthreads = -1;
  depthlimit = 99999;
  disabled = false;
  initialized = false;
  pr_has_been_called = false;
  verbose = false;
  percent = false;
  dopr_preamble = true;
  dopr_threadsort = true;
  dopr_multparent = true;
  dopr_collision = true;
  ref_gettimeofday = -1;
  ref_clock_gettime = -1;
#ifdef _AIX
  ref_read_real_time = -1;
#endif
  funcidx = 0;
#ifdef HAVE_NANOTIME
  cpumhz= 0;
  cyc2sec = -1;
#endif
  tablesize = DEFAULT_TABLE_SIZE;
  tablesizem1 = tablesize - 1;

  GPTLfinalize_gpu<<<1,1>>>();
  return 0;
}

/*
** GPTLstart_instr: start a timer (auto-instrumented)
**
** Input arguments:
**   self: function address
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLstart_instr (void *self)
{
  Timer *ptr;              /* linked list pointer */
  int t;                   /* thread index (of this thread) */
  unsigned int indx;       /* hash table index */
  static const char *thisfunc = "GPTLstart_instr";
  
  if (disabled)
    return 0;
  
  if ( ! initialized)
    return GPTLerror ("%s self=%p: GPTLinitialize has not been called\n", thisfunc, self);

  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /* If current depth exceeds a user-specified limit for print, just increment and return */
  if (stackidx[t].val >= depthlimit) {
    ++stackidx[t].val;
    return 0;
  }

  ptr = getentry_instr (hashtable[t], self, &indx);

  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr && ptr->onflg) {
    ++ptr->recurselvl;
    return 0;
  }

  /*
  ** Increment stackidx[t] unconditionally. This is necessary to ensure the correct
  ** behavior when GPTLstop_instr decrements stackidx[t] unconditionally.
  */
  if (++stackidx[t].val > MAX_STACK-1)
    return GPTLerror ("%s: stack too big\n", thisfunc);

  if ( ! ptr) {     /* Add a new entry and initialize */
    ptr = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (ptr, 0, sizeof (Timer));

    /*
    ** Need to save the address string for later conversion back to a real
    ** name by an offline tool.
    */
    snprintf (ptr->name, MAX_CHARS+1, "%lx", (unsigned long) self);
    ptr->address = self;

    if (update_ll_hash (ptr, t, indx) != 0)
      return GPTLerror ("%s: update_ll_hash error\n", thisfunc);
  }

  if (update_parent_info (ptr, callstack[t], stackidx[t].val) != 0)
    return GPTLerror ("%s: update_parent_info error\n", thisfunc);

  if (update_ptr (ptr, t) != 0)
    return GPTLerror ("%s: update_ptr error\n", thisfunc);

  return (0);
}  

/*
** GPTLstart: start a timer
**
** Input arguments:
**   name: timer name
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLstart (const char *name)               /* timer name */
{
  Timer *ptr;        /* linked list pointer */
  int t;             /* thread index (of this thread) */
  int numchars;      /* number of characters to copy */
  unsigned int indx; /* hash table index */
  static const char *thisfunc = "GPTLstart";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s name=%s: GPTLinitialize has not been called\n", thisfunc, name);

  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /*
  ** If current depth exceeds a user-specified limit for print, just
  ** increment and return
  */
  if (stackidx[t].val >= depthlimit) {
    ++stackidx[t].val;
    return 0;
  }

  /* ptr will point to the requested timer in the current list, or NULL if this is a new entry */
  indx = genhashidx (name);
  ptr = getentry (hashtable[t], name, indx);

  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr && ptr->onflg) {
    ++ptr->recurselvl;
    return 0;
  }

  /*
  ** Increment stackidx[t] unconditionally. This is necessary to ensure the correct
  ** behavior when GPTLstop decrements stackidx[t] unconditionally.
  */
  if (++stackidx[t].val > MAX_STACK-1)
    return GPTLerror ("%s: stack too big\n", thisfunc);

  if ( ! ptr) { /* Add a new entry and initialize */
    ptr = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (ptr, 0, sizeof (Timer));

    numchars = MIN (strlen (name), MAX_CHARS);
    strncpy (ptr->name, name, numchars);
    ptr->name[numchars] = '\0';

    if (update_ll_hash (ptr, t, indx) != 0)
      return GPTLerror ("%s: update_ll_hash error\n", thisfunc);
  }

  if (update_parent_info (ptr, callstack[t], stackidx[t].val) != 0)
    return GPTLerror ("%s: update_parent_info error\n", thisfunc);

  if (update_ptr (ptr, t) != 0)
    return GPTLerror ("%s: update_ptr error\n", thisfunc);

  return (0);
}

/*
** GPTLinit_handle: Initialize a handle for further use by GPTLstart_handle() and GPTLstop_handle()
**
** Input arguments:
**   name: timer name
**
** Output arguments:
**   handle: hash value corresponding to "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLinit_handle (const char *name,     /* timer name */
			      int *handle)          /* handle (output if input value is zero) */
{
  if (disabled)
    return 0;

  *handle = (int) genhashidx (name);
  return 0;
}

/*
** GPTLstart_handle: start a timer based on a handle
**
** Input arguments:
**   name: timer name (required when on input, handle=0)
**   handle: pointer to timer matching "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLstart_handle (const char *name,  /* timer name */
			       int *handle)       /* handle (output if input value is zero) */
{
  Timer *ptr;                            /* linked list pointer */
  int t;                                 /* thread index (of this thread) */
  int numchars;                          /* number of characters to copy */
  static const char *thisfunc = "GPTLstart_handle";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s name=%s: GPTLinitialize has not been called\n", thisfunc, name);

  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /* If current depth exceeds a user-specified limit for print, just increment and return */
  if (stackidx[t].val >= depthlimit) {
    ++stackidx[t].val;
    return 0;
  }

  /*
  ** If handle is zero on input, generate the hash entry and return it to the user.
  ** Otherwise assume it's a previously generated hash index passed in by the user.
  ** Don't need a critical section here--worst case multiple threads will generate the
  ** same handle and store to the same memory location, and this will only happen once.
  */
  if (*handle == 0) {
    *handle = (int) genhashidx (name);
#ifdef VERBOSE
    printf ("%s: name=%s thread %d generated handle=%d\n", thisfunc, name, t, *handle);
#endif
  } else if ((unsigned int) *handle > tablesizem1) {
    return GPTLerror ("%s: Bad input handle=%u exceeds tablesizem1=%d\n", 
		      thisfunc, (unsigned int) *handle, tablesizem1);
  }

  ptr = getentry (hashtable[t], name, (unsigned int) *handle);
  
  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr && ptr->onflg) {
    ++ptr->recurselvl;
    return 0;
  }

  /*
  ** Increment stackidx[t] unconditionally. This is necessary to ensure the correct
  ** behavior when GPTLstop decrements stackidx[t] unconditionally.
  */
  if (++stackidx[t].val > MAX_STACK-1)
    return GPTLerror ("%s: stack too big\n", thisfunc);

  if ( ! ptr) { /* Add a new entry and initialize */
    ptr = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (ptr, 0, sizeof (Timer));

    numchars = MIN (strlen (name), MAX_CHARS);
    strncpy (ptr->name, name, numchars);
    ptr->name[numchars] = '\0';

    if (update_ll_hash (ptr, t, (unsigned int) *handle) != 0)
      return GPTLerror ("%s: update_ll_hash error\n", thisfunc);
  }

  if (update_parent_info (ptr, callstack[t], stackidx[t].val) != 0)
    return GPTLerror ("%s: update_parent_info error\n", thisfunc);

  if (update_ptr (ptr, t) != 0)
    return GPTLerror ("%s: update_ptr error\n", thisfunc);

  return (0);
}

/*
** update_ll_hash: Update linked list and hash table.
**                 Called by all GPTLstart* routines when there is a new entry
**
** Input arguments:
**   ptr:  pointer to timer
**   t:    thread index
**   indx: hash index
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ static int update_ll_hash (Timer *ptr, int t, unsigned int indx)
{
  int nchars;      /* number of chars */
  int nument;      /* number of entries */
  Timer **eptr;    /* for realloc */

  nchars = strlen (ptr->name);
  if (nchars > max_name_len[t])
    max_name_len[t] = nchars;

  last[t]->next = ptr;
  last[t] = ptr;
  ++hashtable[t][indx].nument;
  nument = hashtable[t][indx].nument;
  
  eptr = (Timer **) realloc (hashtable[t][indx].entries, nument * sizeof (Timer *));
  if ( ! eptr)
    return GPTLerror ("update_ll_hash: realloc error\n");

  hashtable[t][indx].entries           = eptr;
  hashtable[t][indx].entries[nument-1] = ptr;

  return 0;
}

/*
** update_ptr: Update timer contents. Called by GPTLstart, GPTLstart_instr and GPTLstart_handle
**
** Input arguments:
**   ptr:  pointer to timer
**   t:    thread index
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ static inline int update_ptr (Timer *ptr, const int t)
{
  double tp2;    /* time stamp */

  ptr->onflg = true;

  if (cpustats.enabled && get_cpustamp (&ptr->cpu.last_utime, &ptr->cpu.last_stime) < 0)
    return GPTLerror ("update_ptr: get_cpustamp error");
  
  if (wallstats.enabled) {
    tp2 = (*ptr2wtimefunc) ();
    ptr->wall.last = tp2;
  }

  return 0;
}

/*
** update_parent_info: update info about parent, and in the parent about this child
**                     Called by all GPTLstart* routines
**
** Arguments:
**   ptr:  pointer to timer
**   callstackt: callstack for this thread
**   stackidxt:  stack index for this thread
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ static inline int update_parent_info (Timer *ptr, 
					       Timer **callstackt, 
                                      int stackidxt) 
{
  int n;             /* loop index through known parents */
  Timer *pptr;       /* pointer to parent in callstack */
  Timer **pptrtmp;   /* for realloc parent pointer array */
  int nparent;       /* number of parents */
  int *parent_count; /* number of times parent invoked this child */
  static const char *thisfunc = "update_parent_info";

  if ( ! ptr )
    return -1;

  if (stackidxt < 0)
    return GPTLerror ("%s: called with negative stackidx\n", thisfunc);

  callstackt[stackidxt] = ptr;

  /* Bump orphan count if the region has no parent (should never happen since "GPTL_ROOT" added) */
  if (stackidxt == 0) {
    ++ptr->norphan;
    return 0;
  }

  pptr = callstackt[stackidxt-1];

  /* If this parent occurred before, bump its count */
  for (n = 0; n < ptr->nparent; ++n) {
    if (ptr->parent[n] == pptr) {
      ++ptr->parent_count[n];
      break;
    }
  }

  /* If this is a new parent, update info */
  if (n == ptr->nparent) {
    ++ptr->nparent;
    nparent = ptr->nparent;
    pptrtmp = (Timer **) realloc (ptr->parent, nparent * sizeof (Timer *));
    if ( ! pptrtmp)
      return GPTLerror ("%s: realloc error pptrtmp nparent=%d\n", thisfunc, nparent);

    ptr->parent = pptrtmp;
    ptr->parent[nparent-1] = pptr;
    parent_count = (int *) realloc (ptr->parent_count, nparent * sizeof (int));
    if ( ! parent_count)
      return GPTLerror ("%s: realloc error parent_count nparent=%d\n", thisfunc, nparent);

    ptr->parent_count = parent_count;
    ptr->parent_count[nparent-1] = 1;
  }

  return 0;
}

/*
** GPTLstop_instr: stop a timer (auto-instrumented)
**
** Input arguments:
**   self: function address
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLstop_instr (void *self)
{
  double tp1 = 0.0;          /* time stamp */
  Timer *ptr;                /* linked list pointer */
  int t;                     /* thread number for this process */
  unsigned int indx;         /* index into hash table */
  long usr = 0;              /* user time (returned from get_cpustamp) */
  long sys = 0;              /* system time (returned from get_cpustamp) */
  static const char *thisfunc = "GPTLstop_instr";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  /* Get the timestamp */    
  if (wallstats.enabled) {
    tp1 = (*ptr2wtimefunc) ();
  }

  if (cpustats.enabled && get_cpustamp (&usr, &sys) < 0)
    return GPTLerror ("%s: bad return from get_cpustamp\n", thisfunc);

  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /* If current depth exceeds a user-specified limit for print, just decrement and return */
  if (stackidx[t].val > depthlimit) {
    --stackidx[t].val;
    return 0;
  }

  ptr = getentry_instr (hashtable[t], self, &indx);

  if ( ! ptr) 
    return GPTLerror ("%s: timer for %p had not been started.\n", thisfunc, self);

  if ( ! ptr->onflg )
    return GPTLerror ("%s: timer %s was already off.\n", thisfunc, ptr->name);

  ++ptr->count;

  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr->recurselvl > 0) {
    ++ptr->nrecurse;
    --ptr->recurselvl;
    return 0;
  }

  if (update_stats (ptr, tp1, usr, sys, t) != 0)
    return GPTLerror ("%s: error from update_stats\n", thisfunc);

  return 0;
}

/*
** GPTLstop: stop a timer
**
** Input arguments:
**   name: timer name
**
** Return value: 0 (success) or -1 (failure)
*/
__host__ int GPTLstop (const char *name)               /* timer name */
{
  double tp1 = 0.0;          /* time stamp */
  Timer *ptr;                /* linked list pointer */
  int t;                     /* thread number for this process */
  unsigned int indx;         /* index into hash table */
  long usr = 0;              /* user time (returned from get_cpustamp) */
  long sys = 0;              /* system time (returned from get_cpustamp) */
  static const char *thisfunc = "GPTLstop";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  /* Get the timestamp */
    
  if (wallstats.enabled) {
    tp1 = (*ptr2wtimefunc) ();
  }

  if (cpustats.enabled && get_cpustamp (&usr, &sys) < 0)
    return GPTLerror ("%s: get_cpustamp error", thisfunc);

  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /* If current depth exceeds a user-specified limit for print, just decrement and return */
  if (stackidx[t].val > depthlimit) {
    --stackidx[t].val;
    return 0;
  }

  indx = genhashidx (name);
  if (! (ptr = getentry (hashtable[t], name, indx)))
    return GPTLerror ("%s thread %d: timer for %s had not been started.\n", thisfunc, t, name);

  if ( ! ptr->onflg )
    return GPTLerror ("%s: timer %s was already off.\n", thisfunc, ptr->name);

  ++ptr->count;

  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr->recurselvl > 0) {
    ++ptr->nrecurse;
    --ptr->recurselvl;
    return 0;
  }

  if (update_stats (ptr, tp1, usr, sys, t) != 0)
    return GPTLerror ("%s: error from update_stats\n", thisfunc);

  return 0;
}

/*
** GPTLstop_handle: stop a timer based on a handle
**
** Input arguments:
**   name: timer name (used only for diagnostics)
**   handle: pointer to timer
**
** Return value: 0 (success) or -1 (failure)
*/
__host__ int GPTLstop_handle (const char *name,     /* timer name */
			      int *handle)          /* handle */
{
  double tp1 = 0.0;          /* time stamp */
  Timer *ptr;                /* linked list pointer */
  int t;                     /* thread number for this process */
  long usr = 0;              /* user time (returned from get_cpustamp) */
  long sys = 0;              /* system time (returned from get_cpustamp) */
  unsigned int indx;
  static const char *thisfunc = "GPTLstop_handle";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  /* Get the timestamp */
  if (wallstats.enabled) {
    tp1 = (*ptr2wtimefunc) ();
  }

  if (cpustats.enabled && get_cpustamp (&usr, &sys) < 0)
    return GPTLerror (0);

  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /* If current depth exceeds a user-specified limit for print, just decrement and return */
  if (stackidx[t].val > depthlimit) {
    --stackidx[t].val;
    return 0;
  }

  indx = (unsigned int) *handle;
  if (indx == 0 || indx > tablesizem1) 
    return GPTLerror ("%s: bad input handle=%u for timer %s.\n", thisfunc, indx, name);
  
  if ( ! (ptr = getentry (hashtable[t], name, indx)))
    return GPTLerror ("%s: handle=%u has not been set for timer %s.\n", 
		      thisfunc, indx, name);

  if ( ! ptr->onflg )
    return GPTLerror ("%s: timer %s was already off.\n", thisfunc, ptr->name);

  ++ptr->count;

  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr->recurselvl > 0) {
    ++ptr->nrecurse;
    --ptr->recurselvl;
    return 0;
  }

  if (update_stats (ptr, tp1, usr, sys, t) != 0)
    return GPTLerror ("%s: error from update_stats\n", thisfunc);

  return 0;
}

/*
** update_stats: update stats inside ptr. Called by GPTLstop, GPTLstop_instr, 
**               GPTLstop_handle
**
** Input arguments:
**   ptr: pointer to timer
**   tp1: input time stamp
**   usr: user time
**   sys: system time
**   t: thread index
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ static inline int update_stats (Timer *ptr, 
					 const double tp1, 
					 const long usr, 
					 const long sys,
					 const int t)
{
  double delta;      /* difference */
  int bidx;          /* bottom of call stack */
  Timer *bptr;       /* pointer to last entry in call stack */
  static const char *thisfunc = "update_stats";

  ptr->onflg = false;

  if (wallstats.enabled) {
    delta = tp1 - ptr->wall.last;
    ptr->wall.accum += delta;
    ptr->wall.latest = delta;

    if (delta < 0.)
      fprintf (stderr, "GPTL: %s: negative delta=%g\n", thisfunc, delta);

    if (ptr->count == 1) {
      ptr->wall.max = delta;
      ptr->wall.min = delta;
    } else {
      if (delta > ptr->wall.max)
        ptr->wall.max = delta;
      if (delta < ptr->wall.min)
        ptr->wall.min = delta;
    }
  }

  if (cpustats.enabled) {
    ptr->cpu.accum_utime += usr - ptr->cpu.last_utime;
    ptr->cpu.accum_stime += sys - ptr->cpu.last_stime;
    ptr->cpu.last_utime   = usr;
    ptr->cpu.last_stime   = sys;
  }

  /* Verify that the timer being stopped is at the bottom of the call stack */
  bidx = stackidx[t].val;
  bptr = callstack[t][bidx];
  if (ptr != bptr) {
    imperfect_nest = true;
    GPTLwarn ("%s: Got timer=%s expected btm of call stack=%s\n",
	      thisfunc, ptr->name, bptr->name);
  }

  --stackidx[t].val;           /* Pop the callstack */
  if (stackidx[t].val < -1) {
    stackidx[t].val = -1;
    return GPTLerror ("%s: tree depth has become negative.\n", thisfunc);
  }

  return 0;
}

/*
** GPTLenable: enable timers
**
** Return value: 0 (success)
*/
__host__ int GPTLenable (void)
{
  disabled = false;
  GPTLenable_gpu<<<1,1>>>();
  return 0;
}

/*
** GPTLdisable: disable timers
**
** Return value: 0 (success)
*/
int GPTLdisable (void)
{
  disabled = true;
  GPTLdisable_gpu<<<1,1>>>();
  return 0;
}

/*
** GPTLstamp: Compute timestamp of usr, sys, and wallclock time (seconds)
**
** Output arguments:
**   wall: wallclock
**   usr:  user time
**   sys:  system time
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLstamp (double *wall, double *usr, double *sys)
{
  struct tms buf;            /* argument to times */

  if ( ! initialized)
    return GPTLerror ("GPTLstamp: GPTLinitialize has not been called\n");

#ifdef HAVE_TIMES
  *usr = 0;
  *sys = 0;

  if (times (&buf) == -1)
    return GPTLerror ("GPTLstamp: times() failed. Results bogus\n");

  *usr = buf.tms_utime / (double) ticks_per_sec;
  *sys = buf.tms_stime / (double) ticks_per_sec;
#endif
  *wall = (*ptr2wtimefunc) ();
  return 0;
}

/*
** GPTLreset: reset all timers to 0
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLreset (void)
{
  int t;             /* index over threads */
  Timer *ptr;        /* linked list index */
  static const char *thisfunc = "GPTLreset";

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  for (t = 0; t < nthreads; t++) {
    for (ptr = timers[t]; ptr; ptr = ptr->next) {
      ptr->onflg = false;
      ptr->count = 0;
      memset (&ptr->wall, 0, sizeof (ptr->wall));
      memset (&ptr->cpu, 0, sizeof (ptr->cpu));
    }
  }

  GPTLreset_gpu<<<1,1>>>();
  if (verbose)
    printf ("%s: accumulators for all timers set to zero\n", thisfunc);

  return 0;
}

/*
** GPTLreset_timer: reset a timer to 0
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLreset_timer (char *name)
{
  int t;             /* index over threads */
  Timer *ptr;        /* linked list index */
  unsigned int indx; /* hash table index */
  static const char *thisfunc = "GPTLreset_timer";

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  if (get_thread_num () != 0)
    return GPTLerror ("%s: Must be called by the master thread\n", thisfunc);

  indx = genhashidx (name);
  for (t = 0; t < nthreads; ++t) {
    ptr = getentry (hashtable[t], name, indx);
    if (ptr) {
      ptr->onflg = false;
      ptr->count = 0;
      memset (&ptr->wall, 0, sizeof (ptr->wall));
      memset (&ptr->cpu, 0, sizeof (ptr->cpu));
    }
  }
  return 0;
}

/* 
** GPTLpr: Print values of all timers
**
** Input arguments:
**   id: integer to append to string "timing."
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLpr (const int id)   /* output file will be named "timing.<id>" */
{
  char outfile[14];         /* name of output file: timing.xxxxxx */
  static const char *thisfunc = "GPTLpr";

  if (id < 0 || id > 999999)
    return GPTLerror ("%s: bad id=%d for output file. Must be >= 0 and < 1000000\n", thisfunc, id);

  sprintf (outfile, "timing.%d", id);

  if (GPTLpr_file (outfile) != 0)
    return GPTLerror ("%s: Error in GPTLpr_file\n", thisfunc);

  return 0;
}

/* 
** GPTLpr_file: Print values of all timers
**
** Input arguments:
**   outfile: Name of output file to write
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int GPTLpr_file (const char *outfile) /* output file to write */
{
  FILE *fp;                 /* file handle to write to */
  Timer *ptr;               /* walk through master thread linked list */
  Timer *tptr;              /* walk through slave threads linked lists */
  Timer sumstats;           /* sum of same timer stats over threads */
  int n, t;                 /* indices */
  unsigned long totcount;   /* total timer invocations */
  float *sum;               /* sum of overhead values (per thread) */
  float osum;               /* sum of overhead over threads */
  bool found;               /* jump out of loop when name found */
  bool foundany;            /* whether summation print necessary */
  bool first;               /* flag 1st time entry found */
  double self_ohd;          /* estimated library overhead in self timer */
  double parent_ohd;        /* estimated library overhead due to self in parent timer */
  int size, rss, share, text, datastack; /* returned from GPTLget_memusage */

  static const char *thisfunc = "GPTLpr_file";

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize() has not been called\n", thisfunc);

  if ( ! (fp = fopen (outfile, "w")))
    fp = stderr;

  /* Print a warning if GPTLerror() was ever called */
  if (GPTLnum_errors () > 0) {
    fprintf (fp, "WARNING: GPTLerror was called at least once during the run.\n");
    fprintf (fp, "Please examine your output for error messages beginning with GPTL...\n");
  }

  /* Print a warning if imperfect nesting was encountered */
  if (imperfect_nest) {
    fprintf (fp, "WARNING: SOME TIMER CALLS WERE DETECTED TO HAVE IMPERFECT NESTING.\n");
    fprintf (fp, "TIMING RESULTS WILL BE PRINTED WITHOUT INDENTING AND NO PARENT-CHILD\n");
    fprintf (fp, "INDENTING WILL BE DONE.\n");
    fprintf (fp, "ALSO: NO MULTIPLE PARENT INFORMATION WILL BE PRINTED SINCE IT MAY CONTAIN ERRORS\n");
  }

  /* A set of nasty ifdefs to tell important aspects of how GPTL was built */
#ifdef HAVE_NANOTIME
  if (funclist[funcidx].option == GPTLnanotime) {
    fprintf (fp, "Clock rate = %f MHz\n", cpumhz);
    fprintf (fp, "Source of clock rate was %s\n", clock_source);
    if (strcmp (clock_source, "/proc/cpuinfo") == 0) {
      fprintf (fp, "WARNING: The contents of /proc/cpuinfo can change in variable frequency CPUs");
      fprintf (fp, "Therefore the use of nanotime (register read) is not recommended on machines so equipped");
    }
#ifdef BIT64
    fprintf (fp, "  BIT64 was true\n");
#else
    fprintf (fp, "  BIT64 was false\n");
#endif
  }
#endif

#if ( defined THREADED_OMP )
  fprintf (fp, "GPTL was built with THREADED_OMP\n");
#else
  fprintf (fp, "GPTL was built without threading\n");
#endif

#ifdef HAVE_MPI
  fprintf (fp, "HAVE_MPI was true\n");

#ifdef HAVE_COMM_F2C
  fprintf (fp, "  HAVE_COMM_F2C was true\n");
#else
  fprintf (fp, "  HAVE_COMM_F2C was false\n");
#endif

#else
  fprintf (fp, "HAVE_MPI was false\n");
#endif

#ifdef ENABLE_NESTEDOMP
  fprintf (fp, "ENABLE_NESTEDOMP was true\n");
#else
  fprintf (fp, "ENABLE_NESTEDOMP was false\n");
#endif

  fprintf (fp, "Underlying timing routine was %s.\n", funclist[funcidx].name);
  (void) GPTLget_overhead (fp, ptr2wtimefunc, getentry, genhashidx, get_thread_num, 
			   stackidx, callstack, hashtable[0], tablesize, imperfect_nest, 
			   &self_ohd, &parent_ohd);
  if (dopr_preamble) {
    fprintf (fp, "\nIf overhead stats are printed, they are the columns labeled self_OH and parent_OH\n"
	     "self_OH is estimated as 2X the Fortran layer cost (start+stop) plust the cost of \n"
	     "a single call to the underlying timing routine.\n"
	     "parent_OH is the overhead for the named timer which is subsumed into its parent.\n"
	     "It is estimated as the cost of a single GPTLstart()/GPTLstop() pair.\n"
             "Print method was %s.\n", methodstr (method));
    fprintf (fp, "\nIf a \'%%_of\' field is present, it is w.r.t. the first timer for thread 0.\n"
             "A '*' in column 1 below means the timer had multiple parents, though the\n"
             "values printed are for all calls.\n"
             "Further down the listing may be more detailed information about multiple\n"
             "parents. Look for 'Multiple parent info'\n\n");
  }

  /* Print the process size at time of call to GPTLpr_file */
  (void) GPTLget_memusage (&size, &rss, &share, &text, &datastack);
  fprintf (fp, "Process size=%d MB rss=%d MB\n\n", size, rss);

  sum = (float *) GPTLallocate (nthreads * sizeof (float), thisfunc);
  
  for (t = 0; t < nthreads; ++t) {
    print_titles (t, fp);
    /*
    ** Print timing stats. If imperfect nesting was detected, print stats by going through
    ** the linked list and do not indent anything due to the possibility of error.
    ** Otherwise, print call tree and properly indented stats via recursive routine. "-1" 
    ** is flag to avoid printing dummy outermost timer, and initialize the depth.
    */
    if (imperfect_nest) {
      for (ptr = timers[t]->next; ptr; ptr = ptr->next) {
	printstats (ptr, fp, t, 0, false, self_ohd, parent_ohd);
      }
    } else {
      printself_andchildren (timers[t], fp, t, -1, self_ohd, parent_ohd);
    }

    /* 
    ** Sum of self+parent overhead across timers is an estimate of total overhead.
    */
    sum[t]   = 0;
    totcount = 0;
    for (ptr = timers[t]->next; ptr; ptr = ptr->next) {
      sum[t]   += ptr->count * (parent_ohd + self_ohd);
      totcount += ptr->count;
    }
    if (wallstats.enabled && overheadstats.enabled)
      fprintf (fp, "\n");
      fprintf (fp, "Overhead sum = %9.3g wallclock seconds\n", sum[t]);
    if (totcount < PRTHRESH)
      fprintf (fp, "Total calls  = %lu\n", totcount);
    else
      fprintf (fp, "Total calls  = %9.3e\n", (float) totcount);
  }

  /* Print per-name stats for all threads */
  if (dopr_threadsort && nthreads > 1) {
    fprintf (fp, "\nSame stats sorted by timer for threaded regions:\n");
    fprintf (fp, "Thd ");

    for (n = 0; n < max_name_len[0]; ++n) /* longest timer name */
      fprintf (fp, " ");

    fprintf (fp, "Called  Recurse ");

    if (cpustats.enabled)
      fprintf (fp, "%s", cpustats.str);
    if (wallstats.enabled) {
      fprintf (fp, "%s", wallstats.str);
      if (percent && timers[0]->next)
        fprintf (fp, "%%_of_%5.5s ", timers[0]->next->name);
      if (overheadstats.enabled)
        fprintf (fp, "%s", overheadstats.str);
    }

    fprintf (fp, "\n");

    /* Start at next to skip dummy */
    for (ptr = timers[0]->next; ptr; ptr = ptr->next) {      
      /* 
      ** To print sum stats, first create a new timer then copy thread 0
      ** stats into it. then sum using "add", and finally print.
      */
      foundany = false;
      first = true;
      sumstats = *ptr;
      for (t = 1; t < nthreads; ++t) {
        found = false;
        for (tptr = timers[t]->next; tptr && ! found; tptr = tptr->next) {
          if (STRMATCH (ptr->name, tptr->name)) {

            /* Only print thread 0 when this timer found for other threads */
            if (first) {
              first = false;
              fprintf (fp, "%3.3d ", 0);
              printstats (ptr, fp, 0, 0, false, self_ohd, parent_ohd);
            }

            found = true;
            foundany = true;
            fprintf (fp, "%3.3d ", t);
            printstats (tptr, fp, 0, 0, false, self_ohd, parent_ohd);
            add (&sumstats, tptr);
          }
        }
      }

      if (foundany) {
        fprintf (fp, "SUM ");
        printstats (&sumstats, fp, 0, 0, false, self_ohd, parent_ohd);
        fprintf (fp, "\n");
      }
    }

    /* Repeat overhead print in loop over threads */
    if (wallstats.enabled && overheadstats.enabled) {
      osum = 0.;
      for (t = 0; t < nthreads; ++t) {
        fprintf (fp, "OVERHEAD.%3.3d (wallclock seconds) = %9.3g\n", t, sum[t]);
        osum += sum[t];
      }
      fprintf (fp, "OVERHEAD.SUM (wallclock seconds) = %9.3g\n", osum);
    }
  }

  /* 
  ** Print info about timers with multiple parents ONLY if imperfect nesting was not discovered
  */
  if (dopr_multparent && ! imperfect_nest) {
    for (t = 0; t < nthreads; ++t) {
      bool some_multparents = false;   /* thread has entries with multiple parents? */
      for (ptr = timers[t]->next; ptr; ptr = ptr->next) {
        if (ptr->nparent > 1) {
          some_multparents = true;
          break;
        }
      }

      if (some_multparents) {
        fprintf (fp, "\nMultiple parent info for thread %d:\n", t);
        if (dopr_preamble && t == 0) {
          fprintf (fp, "Columns are count and name for the listed child\n"
                   "Rows are each parent, with their common child being the last entry, "
                   "which is indented.\n"
                   "Count next to each parent is the number of times it called the child.\n"
                   "Count next to child is total number of times it was called by the "
                   "listed parents.\n\n");
        }

        for (ptr = timers[t]->next; ptr; ptr = ptr->next)
          if (ptr->nparent > 1)
            print_multparentinfo (fp, ptr);
      }
    }
  }

  /* Print hash table stats */
  if (dopr_collision)
    GPTLprint_hashstats (fp, nthreads, hashtable, tablesize);

  /* Stats on GPTL memory usage */
  GPTLprint_memstats (fp, timers, nthreads, tablesize, maxthreads);

  free (sum);

  // Now retrieve  and print the GPU info
  GPTLprint_gpustats (fp, maxwarps_gpu, maxtimers_gpu, gpu_hz, devnum);

  if (fp != stderr && fclose (fp) != 0)
    fprintf (stderr, "%s: Attempt to close %s failed\n", thisfunc, outfile);

  pr_has_been_called = true;
  return 0;
}

/* 
** print_titles: Print headings to output file. If imperfect nesting was detected, print simply by
**               following the linked list. Otherwise, indent use parent-child relationships.
**
** Input arguments:
**   t: thread number
*/
__host__ static void print_titles (int t, FILE *fp)
{
  int n;
  static const char *thisfunc = "print_titles";
  /*
  ** Construct tree for printing timers in parent/child form. get_max_depth() must be called 
  ** AFTER construct_tree() because it relies on the per-parent children arrays being complete.
  */
  if (imperfect_nest) {
    max_depth[t] = 0;   /* No nesting will be printed since imperfect nesting was detected */
  } else {
    if (construct_tree (timers[t], method) != 0)
      printf ("GPTL: %s: failure from construct_tree: output will be incomplete\n", thisfunc);
    max_depth[t] = get_max_depth (timers[t], 0);
  }

  if (t > 0)
    fprintf (fp, "\n");
  fprintf (fp, "Stats for thread %d:\n", t);

  for (n = 0; n < max_depth[t]+1; ++n)    /* +1 to always indent timer name */
    fprintf (fp, "  ");
  for (n = 0; n < max_name_len[t]; ++n) /* longest timer name */
    fprintf (fp, " ");
  fprintf (fp, "Called  Recurse ");

  /* Print strings for enabled timer types */
  if (cpustats.enabled)
    fprintf (fp, "%s", cpustats.str);
  if (wallstats.enabled) {
    fprintf (fp, "%s", wallstats.str);
    if (percent && timers[0]->next)
      fprintf (fp, "%%_of_%5.5s ", timers[0]->next->name);
    if (overheadstats.enabled)
      fprintf (fp, "%s", overheadstats.str);
  }

  fprintf (fp, "\n");
  return;
}

/* 
** construct_tree: Build the parent->children tree starting with knowledge of
**                 parent list for each child.
**
** Input arguments:
**   method:  method to be used to define the links
**
** Input/Output arguments:
**   timerst: Linked list of timers. "children" array for each timer will be constructed
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ int construct_tree (Timer *timerst, Method method)
{
  Timer *ptr;       /* loop through linked list */
  Timer *pptr = 0;  /* parent (init to NULL to avoid compiler warning) */
  int nparent;      /* number of parents */
  int maxcount;     /* max calls by a single parent */
  int n;            /* loop over nparent */
  static const char *thisfunc = "construct_tree";

  /*
  ** Walk the linked list to build the parent-child tree, using whichever
  ** mechanism is in place. newchild() will prevent loops.
  */
  for (ptr = timerst; ptr; ptr = ptr->next) {
    switch (method) {
    case GPTLfirst_parent:
      if (ptr->nparent > 0) {
        pptr = ptr->parent[0];
        if (newchild (pptr, ptr) != 0);
      }
      break;
    case GPTLlast_parent:
      if (ptr->nparent > 0) {
        nparent = ptr->nparent;
        pptr = ptr->parent[nparent-1];
        if (newchild (pptr, ptr) != 0);
      }
      break;
    case GPTLmost_frequent:
      maxcount = 0;
      for (n = 0; n < ptr->nparent; ++n) {
        if (ptr->parent_count[n] > maxcount) {
          pptr = ptr->parent[n];
          maxcount = ptr->parent_count[n];
        }
      }
      if (maxcount > 0) {   /* not an orphan */
        if (newchild (pptr, ptr) != 0);
      }
      break;
    case GPTLfull_tree:
      for (n = 0; n < ptr->nparent; ++n) {
        pptr = ptr->parent[n];
        if (newchild (pptr, ptr) != 0);
      }
      break;
    default:
      return GPTLerror ("GPTL: %s: method %d is not known\n", thisfunc, method);
    }
  }
  return 0;
}

/* 
** methodstr: Return a pointer to a string which represents the method
**
** Input arguments:
**   method: method type
*/
__host__ static const char *methodstr (Method method)
{
  static const char *first_parent  = "first_parent";
  static const char *last_parent   = "last_parent";
  static const char *most_frequent = "most_frequent";
  static const char *full_tree     = "full_tree";
  static const char *Unknown       = "Unknown";

  if (method == GPTLfirst_parent)
    return first_parent;
  else if (method == GPTLlast_parent)
    return last_parent;
  else if (method == GPTLmost_frequent)
    return most_frequent;
  else if (method == GPTLfull_tree)
    return full_tree;
  else
    return Unknown;
}

/* 
** newchild: Add an entry to the children list of parent. Use function
**   is_descendant() to prevent infinite loops. 
**
** Input arguments:
**   child:  child to be added
**
** Input/output arguments:
**   parent: parent node which will have "child" added to its "children" array
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ static int newchild (Timer *parent, Timer *child)
{
  int nchildren;     /* number of children (temporary) */
  Timer **chptr;     /* array of pointers to children */
  static const char *thisfunc = "newchild";

  if (parent == child)
    return GPTLerror ("%s: child %s can't be a parent of itself\n", thisfunc, child->name);

  /*
  ** To guarantee no loops, ensure that proposed parent isn't already a descendant of 
  ** proposed child
  */
  if (is_descendant (child, parent)) {
    return GPTLerror ("GPTL: %s: loop detected: NOT adding %s to descendant list of %s. "
                      "Proposed parent is in child's descendant path.\n",
                      thisfunc, child->name, parent->name);
  }

  /* 
  ** Add child to parent's array of children if it isn't already there (e.g. by an earlier call
  ** to GPTLpr*)
  */
  if ( ! is_onlist (child, parent)) {
    ++parent->nchildren;
    nchildren = parent->nchildren;
    chptr = (Timer **) realloc (parent->children, nchildren * sizeof (Timer *));
    if ( ! chptr)
      return GPTLerror ("%s: realloc error\n", thisfunc);
    parent->children = chptr;
    parent->children[nchildren-1] = child;
  }

  return 0;
}

/* 
** get_max_depth: Determine the maximum call tree depth by traversing the
**   tree recursively
**
** Input arguments:
**   ptr:        Starting timer
**   startdepth: current depth when function invoked 
**
** Return value: maximum depth
*/
__host__ static int get_max_depth (const Timer *ptr, const int startdepth)
{
  int maxdepth = startdepth;
  int depth;
  int n;

  for (n = 0; n < ptr->nchildren; ++n)
    if ((depth = get_max_depth (ptr->children[n], startdepth+1)) > maxdepth)
      maxdepth = depth;

  return maxdepth;
}

/* 
** is_descendant: Determine whether node2 is in the descendant list for
**   node1
**
** Input arguments:
**   node1: starting node for recursive search
**   node2: node to be searched for
**
** Return value: true or false
*/
__host__ static int is_descendant (const Timer *node1, const Timer *node2)
{
  int n;

  /* Breadth before depth for efficiency */
  for (n = 0; n < node1->nchildren; ++n)
    if (node1->children[n] == node2)
      return 1;

  for (n = 0; n < node1->nchildren; ++n)
    if (is_descendant (node1->children[n], node2))
      return 1;

  return 0;
}

/* 
** is_onlist: Determine whether child is in parent's list of children
**
** Input arguments:
**   child: who to search for
**   parent: search through his list of children
**
** Return value: true or false
*/
__host__ static int is_onlist (const Timer *child, const Timer *parent)
{
  int n;

  for (n = 0; n < parent->nchildren; ++n) {
    if (child == parent->children[n])
      return 1;
  }

  return 0;
}

/* 
** printstats: print a single timer
**
** Input arguments:
**   timer:        timer for which to print stats
**   fp:           file descriptor to write to
**   t:            thread number
**   depth:        depth to indent timer
**   doindent:     whether indenting will be done
**   tot_overhead: underlying timing routine overhead
*/
__host__ static void printstats (const Timer *timer,
				 FILE *fp,
				 int t,
				 int depth,
				 bool doindent,
				 double self_ohd,
				 double parent_ohd)
{
  int i;               /* index */
  int indent;          /* index for indenting */
  int extraspace;      /* for padding to length of longest name */
  float fusr;          /* user time as float */
  float fsys;          /* system time as float */
  float usrsys;        /* usr + sys */
  float elapse;        /* elapsed time */
  float wallmax;       /* max wall time */
  float wallmin;       /* min wall time */
  float ratio;         /* percentage calc */
  static const char *thisfunc = "printstats";

  if (timer->onflg && verbose)
    fprintf (stderr, "GPTL: %s: timer %s had not been turned off\n", thisfunc, timer->name);

  /* Flag regions having multiple parents with a "*" in column 1 */
  if (doindent) {
    if (timer->nparent > 1)
      fprintf (fp, "* ");
    else
      fprintf (fp, "  ");

    /* Indent to depth of this timer */
    for (indent = 0; indent < depth; ++indent)
      fprintf (fp, "  ");
  }

  fprintf (fp, "%s", timer->name);

  /* Pad to length of longest name */
  extraspace = max_name_len[t] - strlen (timer->name);
  for (i = 0; i < extraspace; ++i)
    fprintf (fp, " ");

  /* Pad to max indent level */
  if (doindent)
    for (indent = depth; indent < max_depth[t]; ++indent)
      fprintf (fp, "  ");

  /* 
  ** Don't print stats if the timer is currently on: too dangerous since the timer needs 
  ** to be stopped to have currently accurate timings
  */
  if (timer->onflg) {
    fprintf (fp, " NOT PRINTED: timer is currently ON\n");
    return;
  }

  if (timer->count < PRTHRESH) {
    if (timer->nrecurse > 0)
      fprintf (fp, "%8lu %6lu ", timer->count, timer->nrecurse);
    else
      fprintf (fp, "%8lu    -   ", timer->count);
  } else {
    if (timer->nrecurse > 0)
      fprintf (fp, "%8.1e %6.0e ", (float) timer->count, (float) timer->nrecurse);
    else
      fprintf (fp, "%8.1e    -   ", (float) timer->count);
  }

  if (cpustats.enabled) {
    fusr = timer->cpu.accum_utime / (float) ticks_per_sec;
    fsys = timer->cpu.accum_stime / (float) ticks_per_sec;
    usrsys = fusr + fsys;
    fprintf (fp, "%9.3f %9.3f %9.3f ", fusr, fsys, usrsys);
  }

  if (wallstats.enabled) {
    elapse = timer->wall.accum;
    wallmax = timer->wall.max;
    wallmin = timer->wall.min;

    if (elapse < 0.01)
      fprintf (fp, "%9.2e ", elapse);
    else
      fprintf (fp, "%9.3f ", elapse);

    if (wallmax < 0.01)
      fprintf (fp, "%9.2e ", wallmax);
    else
      fprintf (fp, "%9.3f ", wallmax);

    if (wallmin < 0.01)
      fprintf (fp, "%9.2e ", wallmin);
    else
      fprintf (fp, "%9.3f ", wallmin);

    if (percent && timers[0]->next) {
      ratio = 0.;
      if (timers[0]->next->wall.accum > 0.)
        ratio = (timer->wall.accum * 100.) / timers[0]->next->wall.accum;
      fprintf (fp, " %9.2f ", ratio);
    }

    if (overheadstats.enabled) {
      fprintf (fp, "%9.3f %9.3f ", timer->count*self_ohd, timer->count*parent_ohd);
    }
  }

  fprintf (fp, "\n");
}

/* 
** print_multparentinfo: 
**
** Input arguments:
** Input/output arguments:
*/
__host__ void print_multparentinfo (FILE *fp, 
				    Timer *ptr)
{
  int n;

  if (ptr->norphan > 0) {
    if (ptr->norphan < PRTHRESH)
      fprintf (fp, "%8u %-32s\n", ptr->norphan, "ORPHAN");
    else
      fprintf (fp, "%8.1e %-32s\n", (float) ptr->norphan, "ORPHAN");
  }

  for (n = 0; n < ptr->nparent; ++n) {
    if (ptr->parent_count[n] < PRTHRESH)
      fprintf (fp, "%8d %-32s\n", ptr->parent_count[n], ptr->parent[n]->name);
    else
      fprintf (fp, "%8.1e %-32s\n", (float) ptr->parent_count[n], ptr->parent[n]->name);
  }

  if (ptr->count < PRTHRESH)
    fprintf (fp, "%8lu   %-32s\n\n", ptr->count, ptr->name);
  else
    fprintf (fp, "%8.1e   %-32s\n\n", (float) ptr->count, ptr->name);
}

/* 
** add: add the contents of tin to tout
**
** Input arguments:
**   tin:  input timer
** Input/output arguments:
**   tout: output timer summed into
*/
__host__ static void add (Timer *tout,   
			  const Timer *tin)
{
  tout->count += tin->count;

  if (wallstats.enabled) {
    tout->wall.accum += tin->wall.accum;
    
    tout->wall.max = MAX (tout->wall.max, tin->wall.max);
    tout->wall.min = MIN (tout->wall.min, tin->wall.min);
  }

  if (cpustats.enabled) {
    tout->cpu.accum_utime += tin->cpu.accum_utime;
    tout->cpu.accum_stime += tin->cpu.accum_stime;
  }
}

#ifdef HAVE_MPI

/* 
** GPTLbarrier: When MPI enabled, set and time an MPI barrier
**
** Input arguments:
**   comm: commuicator (e.g. MPI_COMM_WORLD). If zero, use MPI_COMM_WORLD
**   name: region name
**
** Return value: 0 (success)
*/
int GPTLbarrier (MPI_Comm comm, const char *name)
{
  int ret;
  static const char *thisfunc = "GPTLbarrier";

  ret = GPTLstart (name);
  if ((ret = MPI_Barrier (comm)) != MPI_SUCCESS)
    return GPTLerror ("%s: Bad return from MPI_Barrier=%d", thisfunc, ret);
  ret = GPTLstop (name);
  return 0;
}
#endif    /* HAVE_MPI */

/*
** get_cpustamp: Invoke the proper system timer and return stats.
**
** Output arguments:
**   usr: user time
**   sys: system time
**
** Return value: 0 (success)
*/
__host__ static inline int get_cpustamp (long *usr, long *sys)
{
#ifdef HAVE_TIMES
  struct tms buf;

  (void) times (&buf);
  *usr = buf.tms_utime;
  *sys = buf.tms_stime;
  return 0;
#else
  return GPTLerror ("GPTL: get_cpustamp: times() not available\n");
#endif
}

/*
** GPTLquery: return current status info about a timer. If certain stats are not 
** enabled, they should just have zeros in them.
** 
** Input args:
**   name:        timer name
**   t:           thread number (if < 0, the request is for the current thread)
**
** Output args:
**   count:            number of times this timer was called
**   onflg:            whether timer is currently on
**   wallclock:        accumulated wallclock time
**   usr:              accumulated user CPU time
**   sys:              accumulated system CPU time
*/
__host__ int GPTLquery (const char *name, 
			int t,
			int *count,
			int *onflg,
			double *wallclock,
			double *dusr,
			double *dsys)
{
  Timer *ptr;                /* linked list pointer */
  unsigned int indx;         /* linked list index returned from getentry (unused) */
  static const char *thisfunc = "GPTLquery";
  
  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);
  
  /* If t is < 0, assume the request is for the current thread */
  if (t < 0) {
    if ((t = get_thread_num ()) < 0)
      return GPTLerror ("%s: get_thread_num failure\n", thisfunc);
  } else {
    if (t >= maxthreads)
      return GPTLerror ("%s: requested thread %d is too big\n", thisfunc, t);
  }

  indx = genhashidx (name);
  ptr = getentry (hashtable[t], name, indx);
  if ( !ptr)
    return GPTLerror ("%s: requested timer %s does not have a name hash\n", thisfunc, name);

  *onflg     = ptr->onflg;
  *count     = ptr->count;
  *wallclock = ptr->wall.accum;
  *dusr      = ptr->cpu.accum_utime / (double) ticks_per_sec;
  *dsys      = ptr->cpu.accum_stime / (double) ticks_per_sec;
  return 0;
}

/*
** GPTLget_wallclock: return wallclock accumulation for a timer.
** 
** Input args:
**   timername: timer name
**   t:         thread number (if < 0, the request is for the current thread)
**
** Output args:
**   value: current wallclock accumulation for the timer
*/
__host__ int GPTLget_wallclock (const char *timername,
				int t,
				double *value)
{
  void *self;          /* timer address when hash entry generated with *_instr */
  Timer *ptr;          /* linked list pointer */
  unsigned int indx;   /* hash index returned from getentry (unused) */
  static const char *thisfunc = "GPTLget_wallclock";
  
  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  if ( ! wallstats.enabled)
    return GPTLerror ("%s: wallstats not enabled\n", thisfunc);
  
  /* If t is < 0, assume the request is for the current thread */
  if (t < 0) {
    if ((t = get_thread_num ()) < 0)
      return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);
  } else {
    if (t >= maxthreads)
      return GPTLerror ("%s: requested thread %d is too big\n", thisfunc, t);
  }
  
  /* 
  ** Don't know whether hashtable entry for timername was generated with 
  ** *_instr() or not, so try both possibilities
  */
  indx = genhashidx (timername);
  ptr = getentry (hashtable[t], timername, indx);
  if ( !ptr) {
    if (sscanf (timername, "%lx", (unsigned long *) &self) < 1)
      return GPTLerror ("%s: requested timer %s does not exist\n", thisfunc, timername);
    ptr = getentry_instr (hashtable[t], self, &indx);
    if ( !ptr)
      return GPTLerror ("%s: requested timer %s does not exist\n", thisfunc, timername);
  }
  *value = ptr->wall.accum;
  return 0;
}

/*
** GPTLget_wallclock_latest: return most recent wallclock value for a timer.
** 
** Input args:
**   timername: timer name
**   t:         thread number (if < 0, the request is for the current thread)
**
** Output args:
**   value: most recent wallclock value for the timer
*/
__host__ int GPTLget_wallclock_latest (const char *timername,
				       int t,
				       double *value)
{
  Timer *ptr;          /* linked list pointer */
  unsigned int indx;   /* hash index returned from getentry (unused) */
  static const char *thisfunc = "GPTLget_wallclock_latest";
  
  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  if ( ! wallstats.enabled)
    return GPTLerror ("%s: wallstats not enabled\n", thisfunc);
  
  /* If t is < 0, assume the request is for the current thread */
  if (t < 0) {
    if ((t = get_thread_num ()) < 0)
      return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);
  } else {
    if (t >= maxthreads)
      return GPTLerror ("%s: requested thread %d is too big\n", thisfunc, t);
  }
  
  indx = genhashidx (timername);
  ptr = getentry (hashtable[t], timername, indx);
  if ( !ptr)
    return GPTLerror ("%s: requested timer %s does not exist\n", thisfunc, timername);
  *value = ptr->wall.latest;
  return 0;
}

/*
** GPTLget_threadwork: For a timer, across threads compute max work and imbalance
**
** Input arguments:
**   name: timer name
**
** Output arguments:
**   maxwork: maximum work across threads
**   imbal:   imbalance vs. perfectly distributed workload
**
** Return value: 0 (success) or -1 (failure)
*/
__host__ int GPTLget_threadwork (const char *name, 
				 double *maxwork,
				 double *imbal)
{
  Timer *ptr;                  /* linked list pointer */
  int t;                       /* thread number for this process */
  int nfound = 0;              /* number of threads which did work (must be > 0 */
  unsigned int indx;           /* index into hash table */
  double innermax = 0.;        /* maximum work across threads */
  double totalwork = 0.;       /* total work done by all threads */
  double balancedwork;         /* time if work were perfectly load balanced */
  static const char *thisfunc = "GPTLget_threadwork";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  if ( ! wallstats.enabled)
    return GPTLerror ("%s: wallstats must be enabled to call this function\n", thisfunc);

  if (get_thread_num () != 0)
    return GPTLerror ("%s: Must be called by the master thread\n", thisfunc);

  indx = genhashidx (name);
  for (t = 0; t < nthreads; ++t) {
    ptr = getentry (hashtable[t], name, indx);
    if (ptr) {
      ++nfound;
      innermax = MAX (innermax, ptr->wall.accum);
      totalwork += ptr->wall.accum;
    }
  }

  /* It's an error to call this routine for a region that does not exist */
  if (nfound == 0)
    return GPTLerror ("%s: No entries exist for name=%s\n", thisfunc, name);

  /*
  ** A perfectly load-balanced calculation would take time=totalwork/nthreads
  ** Therefore imbalance is slowest thread time minus this number
  */
  balancedwork = totalwork / nthreads;
  *maxwork = innermax;
  *imbal = innermax - balancedwork;

  return 0;
}

/*
** GPTLstartstop_val: Take user input to treat as the result of calling start/stop
**
** Input arguments:
**   name: timer name
**   value: value to add to the timer
**
** Return value: 0 (success) or -1 (failure)
*/
__host__ int GPTLstartstop_val (const char *name, 
				double value)
{
  Timer *ptr;                /* linked list pointer */
  int t;                     /* thread number for this process */
  unsigned int indx;         /* index into hash table */
  static const char *thisfunc = "GPTLstartstop_val";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  if ( ! wallstats.enabled)
    return GPTLerror ("%s: wallstats must be enabled to call this function\n", thisfunc);

  if (value < 0.)
    return GPTLerror ("%s: Input value must not be negative\n", thisfunc);

  /* getentry requires the thread number */
  if ((t = get_thread_num ()) < 0)
    return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);

  /* Find out if the timer already exists */
  indx = genhashidx (name);
  ptr = getentry (hashtable[t], name, indx);

  if (ptr) {
    /*
    ** The timer already exists. Bump the count manually, update the time stamp,
    ** and let control jump to the point where wallclock settings are adjusted.
    */
    ++ptr->count;
    ptr->wall.last = (*ptr2wtimefunc) ();
  } else {
    /*
    ** Need to call start/stop to set up linked list and hash table.
    ** "count" and "last" will also be set properly by the call to this pair.
    */
    if (GPTLstart (name) != 0)
      return GPTLerror ("%s: Error from GPTLstart\n", thisfunc);

    if (GPTLstop (name) != 0)
      return GPTLerror ("%s: Error from GPTLstop\n", thisfunc);

    /* start/stop pair just called should guarantee ptr will be found */
    if ( ! (ptr = getentry (hashtable[t], name, indx)))
      return GPTLerror ("%s: Unexpected error from getentry\n", thisfunc);

    ptr->wall.min = value; /* Since this is the first call, set min to user input */
    /* 
    ** Minor mod: Subtract the overhead of the above start/stop call, before
    ** adding user input
    */
    ptr->wall.accum -= ptr->wall.latest;
  }

  /* Overwrite the values with user input */
  ptr->wall.accum += value;
  ptr->wall.latest = value;
  if (value > ptr->wall.max)
    ptr->wall.max = value;

  /* On first call this setting is unnecessary but avoid an "if" test for efficiency */
  if (value < ptr->wall.min)
    ptr->wall.min = value;

  return 0;
}

/*
** GPTLget_count: return number of start/stop calls for a timer.
** 
** Input args:
**   timername: timer name
**   t:         thread number (if < 0, the request is for the current thread)
**
** Output args:
**   count: current number of start/stop calls for the timer
*/
__host__ int GPTLget_count (const char *timername,
			    int t,
			    int *count)
{
  void *self;          /* timer address when hash entry generated with *_instr */
  Timer *ptr;          /* linked list pointer */
  unsigned int indx;   /* hash index returned from getentry (unused) */
  static const char *thisfunc = "GPTLget_count";
  
  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);

  /* If t is < 0, assume the request is for the current thread */
  if (t < 0) {
    if ((t = get_thread_num ()) < 0)
      return GPTLerror ("%s: bad return from get_thread_num\n", thisfunc);
  } else {
    if (t >= maxthreads)
      return GPTLerror ("%s: requested thread %d is too big\n", thisfunc, t);
  }
  
  /* 
  ** Don't know whether hashtable entry for timername was generated with 
  ** *_instr() or not, so try both possibilities
  */
  indx = genhashidx (timername);
  ptr = getentry (hashtable[t], timername, indx);
  if ( !ptr) {
    if (sscanf (timername, "%lx", (unsigned long *) &self) < 1)
      return GPTLerror ("%s: requested timer %s does not exist\n", thisfunc, timername);
    ptr = getentry_instr (hashtable[t], self, &indx);
    if ( !ptr)
      return GPTLerror ("%s: requested timer %s does not exist\n", thisfunc, timername);
  }
  *count = ptr->count;
  return 0;
}

/*
** GPTLget_nregions: return number of regions (i.e. timer names) for this thread
** 
** Input args:
**   t:    thread number (if < 0, the request is for the current thread)
**
** Output args:
**   nregions: number of regions
*/
__host__ int GPTLget_nregions (int t, 
			       int *nregions)
{
  Timer *ptr;     /* walk through linked list */
  static const char *thisfunc = "GPTLget_nregions";

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);
  
  /*
  ** If t is < 0, assume the request is for the current thread
  */
  
  if (t < 0) {
    if ((t = get_thread_num ()) < 0)
      return GPTLerror ("%s: get_thread_num failure\n", thisfunc);
  } else {
    if (t >= maxthreads)
      return GPTLerror ("%s: requested thread %d is too big\n", thisfunc, t);
  }
  
  *nregions = 0;
  for (ptr = timers[t]->next; ptr; ptr = ptr->next) 
    ++*nregions;

  return 0;
}

/*
** GPTLget_regionname: return region name for this thread
** 
** Input args:
**   t:      thread number (if < 0, the request is for the current thread)
**   region: region number
**   nc:     max number of chars to put in name
**
** Output args:
**   name    region name
*/
__host__ int GPTLget_regionname (int t,      /* thread number */
				 int region, /* region number (0-based) */
				 char *name, /* output region name */
				 int nc)     /* number of chars in name (free form Fortran) */
{
  int ncpy;    /* number of characters to copy */
  int i;       /* index */
  Timer *ptr;  /* walk through linked list */
  static const char *thisfunc = "GPTLget_regionname";

  if ( ! initialized)
    return GPTLerror ("%s: GPTLinitialize has not been called\n", thisfunc);
  
  // If t is < 0, assume the request is for the current thread
  if (t < 0) {
    if ((t = get_thread_num ()) < 0)
      return GPTLerror ("%s: get_thread_num failure\n", thisfunc);
  } else {
    if (t >= maxthreads)
      return GPTLerror ("%s: requested thread %d is too big\n", thisfunc, t);
  }
  
  ptr = timers[t]->next;
  for (i = 0; i < region; i++) {
    if ( ! ptr)
      return GPTLerror ("%s: timer number %d does not exist in thread %d\n", thisfunc, region, t);
    ptr = ptr->next;
  }

  if (ptr) {
    ncpy = MIN (nc, strlen (ptr->name));
    strncpy (name, ptr->name, ncpy);
    
    /* Adding the \0 is only important when called from C */
    if (ncpy < nc)
      name[ncpy] = '\0';
  } else {
    return GPTLerror ("%s: timer number %d does not exist in thread %d\n", thisfunc, region, t);
  }
  return 0;
}

/*
** GPTLis_initialized: Return whether GPTL has been initialized
*/
__host__ int GPTLis_initialized (void)
{
  return (int) initialized;
}

/*
** getentry_instr: find hash table entry and return a pointer to it
**
** Input args:
**   hashtable: the hashtable (array)
**   self:      input address (from -finstrument-functions)
** Output args:
**   indx:      hashtable index
**
** Return value: pointer to the entry, or NULL if not found
*/
__host__ static inline Timer *getentry_instr (const Hashentry *hashtable, /* hash table */
					      void *self,                 /* address */
					      unsigned int *indx)         /* hash index */
{
  int i;
  Timer *ptr = 0;  /* return value when entry not found */

  /*
  ** Hash index is timer address modulo the table size
  ** On most machines, right-shifting the address helps because linkers often
  ** align functions on even boundaries
  */
  *indx = (((unsigned long) self) >> 4) % tablesize;
  for (i = 0; i < hashtable[*indx].nument; ++i) {
    if (hashtable[*indx].entries[i]->address == self) {
      ptr = hashtable[*indx].entries[i];
      break;
    }
  }
  return ptr;
}

/*
** genhashidx: generate hash index
**
** Input args:
**   name: string to be hashed on
**
** Return value: hash value
*/
#define NEWWAY
__host__ static inline unsigned int genhashidx (const char *name)
{
  const unsigned char *c;       /* pointer to elements of "name" */
  unsigned int indx;            /* return value of function */
#ifdef NEWWAY
  unsigned int mididx, lastidx; /* mid and final index of name */

  lastidx = strlen (name) - 1;
  mididx = lastidx / 2;
#else
  int i;                        /* iterator (OLDWAY only) */
#endif
  /* 
  ** Disallow a hash index of zero (by adding 1 at the end) since user input of an uninitialized 
  ** value, though an error, has a likelihood to be zero.
  */
#ifdef NEWWAY
  c = (unsigned char *) name;
  indx = (MAX_CHARS*c[0] + (MAX_CHARS-mididx)*c[mididx] + (MAX_CHARS-lastidx)*c[lastidx]) % tablesizem1 + 1;
#else
  indx = 0;
  i = MAX_CHARS;
#pragma unroll(2)
  for (c = (unsigned char *) name; *c && i > 0; ++c) {
    indx += i*(*c);
    --i;
  }
  indx = indx % tablesizem1 + 1;
#endif

  return indx;
}

/*
** getentry: find the entry in the hash table and return a pointer to it.
**
** Input args:
**   hashtable: the hashtable (array)
**   indx:      hashtable index
**
** Return value: pointer to the entry, or NULL if not found
*/
__host__ static inline Timer *getentry (const Hashentry *hashtable, /* hash table */
					const char *name,           /* name to hash */
					unsigned int indx)          /* hash index */
{
  int i;                      /* loop index */
  Timer *ptr = 0;             /* return value when entry not found */

  /* 
  ** If nument exceeds 1 there was one or more hash collisions and we must search
  ** linearly through the array of names with the same hash for a match
  */
#pragma novector
  for (i = 0; i < hashtable[indx].nument; i++) {
    if (STRMATCH (name, hashtable[indx].entries[i]->name)) {
      ptr = hashtable[indx].entries[i];
      break;
    }
  }
  return ptr;
}

/*
** Add entry points for auto-instrumented codes
** Auto instrumentation flags for various compilers:
**
** gcc, pathcc, icc: -finstrument-functions
** pgcc:             -Minstrument:functions
** xlc:              -qdebug=function_trace
*/

#ifdef _AIX
void __func_trace_enter (const char *function_name,
                         const char *file_name,
                         int line_number,
                         void **const user_data)
{
  char msg[MSGSIZ];
  int size, rss, share, text, datastack;
  int world_iam;
#ifdef HAVE_MPI
  int flag = 0;
  int ret;
#endif

  if (dopr_memusage && get_thread_num() == 0) {
    (void) GPTLget_memusage (&size, &rss, &share, &text, &datastack);
    if (rss > rssmax) {
      rssmax = rss;
      world_iam = 0;
#ifdef HAVE_MPI
      ret = MPI_Initialized (&flag);
      if (ret == MPI_SUCCESS && flag) 
	ret = MPI_Comm_rank (MPI_COMM_WORLD, &world_iam);
#endif
      snprintf (msg, MSGSIZ, "world_iam=%d begin %s rss grew", world_iam, function_name);
      (void) GPTLprint_memusage (msg);
    }
  }
  (void) GPTLstart (function_name);
}
  
__host__ void __func_trace_exit (const char *function_name,
				 const char *file_name,
				 int line_number,
				 void **const user_data)
{
  char msg[MSGSIZ];
  int size, rss, share, text, datastack;
  int world_iam;
#ifdef HAVE_MPI
  int flag = 0;
  int ret;
#endif

  (void) GPTLstop (function_name);

  if (dopr_memusage && get_thread_num() == 0) {
    (void) GPTLget_memusage (&size, &rss, &share, &text, &datastack);
    if (rss > rssmax) {
      rssmax = rss;
      world_iam = 0;
#ifdef HAVE_MPI
      ret = MPI_Initialized (&flag);
      if (ret == MPI_SUCCESS && flag) 
	ret = MPI_Comm_rank (MPI_COMM_WORLD, &world_iam);
#endif
      snprintf (msg, MSGSIZ, "world_iam=%d end %s rss grew", world_iam, function_name);
      (void) GPTLprint_memusage (msg);
    }
  }
}
  
#else
//_AIX not defined
  
__host__ void __cyg_profile_func_enter (void *this_fn,
					void *call_site)
{
#ifdef HAVE_BACKTRACE
  void *buffer[2];
  int nptrs;
  char **strings;
#endif
  char msg[MSGSIZ];
  int size, rss, share, text, datastack;
  int world_iam;
#ifdef HAVE_MPI
  int flag = 0;
  int ret;
#endif

  if (dopr_memusage && get_thread_num() == 0) {
    (void) GPTLget_memusage (&size, &rss, &share, &text, &datastack);
    if (rss > rssmax) {
      rssmax = rss;
      world_iam = 0;
#ifdef HAVE_MPI
      ret = MPI_Initialized (&flag);
      if (ret == MPI_SUCCESS && flag) 
	ret = MPI_Comm_rank (MPI_COMM_WORLD, &world_iam);
#endif

#ifdef HAVE_BACKTRACE
      nptrs = backtrace (buffer, 2);
      strings = backtrace_symbols (buffer, nptrs);
      snprintf (msg, MSGSIZ, "world_iam=%d begin %s rss grew", world_iam, strings[1]);
      free (strings);  /* needed because backtrace_symbols allocated the space */
#else
      snprintf (msg, MSGSIZ, "world_iam=%d begin %lx rss grew", world_iam, (unsigned long) this_fn);
#endif
      (void) GPTLprint_memusage (msg);
    }
  }
  (void) GPTLstart_instr (this_fn);
}

__host__ void __cyg_profile_func_exit (void *this_fn,
				       void *call_site)
{
#ifdef HAVE_BACKTRACE
  void *buffer[2];
  int nptrs;
  char **strings;
#endif
  char msg[MSGSIZ];
  int size, rss, share, text, datastack;
  int world_iam;
#ifdef HAVE_MPI
  int flag = 0;
  int ret;
#endif

  (void) GPTLstop_instr (this_fn);

  if (dopr_memusage && get_thread_num() == 0) {
    (void) GPTLget_memusage (&size, &rss, &share, &text, &datastack);
    if (rss > rssmax) {
      rssmax = rss;
      world_iam = 0;
#ifdef HAVE_MPI
      ret = MPI_Initialized (&flag);
      if (ret == MPI_SUCCESS && flag) 
	ret = MPI_Comm_rank (MPI_COMM_WORLD, &world_iam);
#endif
#ifdef HAVE_BACKTRACE
      nptrs = backtrace (buffer, 2);
      strings = backtrace_symbols (buffer, nptrs);
      snprintf (msg, MSGSIZ, "world_iam=%d end %s rss grew", world_iam, (char *) strings[1]);
      free (strings);  /* needed because backtrace_symbols allocated the space */
#else
      snprintf (msg, MSGSIZ, "world_iam=%d end %lx rss grew", world_iam, (unsigned long) this_fn);
#endif
      (void) GPTLprint_memusage (msg);
    }
  }
}
#endif
// _AIX false branch

#ifdef HAVE_NANOTIME
// Copied from PAPI library
__host__ static inline long long nanotime (void)
{
  long long val = 0;
#ifdef BIT64
  do {
    unsigned int a, d;
    asm volatile ("rdtsc":"=a" (a), "=d" (d));
    (val) = ((long long) a) | (((long long) d) << 32);
  } while (0);
#else
  __asm__ __volatile__("rdtsc":"=A" (val): );
#endif
  return val;
}

#define LEN 4096

__host__ static float get_clockfreq ()
{
  FILE *fd = 0;
  char buf[LEN];
  int is;
  float freq = -1.;             /* clock frequency (MHz) */
  static const char *thisfunc = "get_clockfreq";
  static char *max_freq_fn = "/sys/devices/system/cpu/cpu0/cpufreq/cpuinfo_max_freq";
  static char *cpuinfo_fn = "/proc/cpuinfo";

  /* First look for max_freq, but that isn't guaranteed to exist */

  if ((fd = fopen (max_freq_fn, "r"))) {
    if (fgets (buf, LEN, fd)) {
      freq = 0.001 * (float) atof (buf);  /* Convert from KHz to MHz */
      if (verbose)
        printf ("GPTL: %s: Using max clock freq = %f for timing\n", thisfunc, freq);
      (void) fclose (fd);
      clock_source = max_freq_fn;
      return freq;
    } else {
      (void) fclose (fd);
    }
  }

  /* 
  ** Next try /proc/cpuinfo. That has the disadvantage that it may give wrong info
  ** for processors that have either idle or turbo mode
  */
  if (verbose && freq < 0.)
    printf ("GPTL: %s: CAUTION: Can't find max clock freq. Trying %s instead\n",
            thisfunc, cpuinfo_fn);

  if ( ! (fd = fopen (cpuinfo_fn, "r"))) {
    fprintf (stderr, "GPTL: %s: can't open %s\n", thisfunc, cpuinfo_fn);
    return -1.;
  }

  while (fgets (buf, LEN, fd)) {
    if (strncmp (buf, "cpu MHz", 7) == 0) {
      for (is = 7; buf[is] != '\0' && !isdigit (buf[is]); is++);
      if (isdigit (buf[is])) {
        freq = (float) atof (&buf[is]);
        (void) fclose (fd);
        clock_source = cpuinfo_fn;
        return freq;
      }
    }
  }

  (void) fclose (fd);
  return -1.;
}
#endif

/*
** The following are the set of underlying timing routines which may or may
** not be available. And their accompanying init routines.
** NANOTIME is currently only available on x86.
*/
__host__ static int init_nanotime ()
{
  static const char *thisfunc = "init_nanotime";
#ifdef HAVE_NANOTIME
  if ((cpumhz = get_clockfreq ()) < 0)
    return GPTLerror ("%s: Can't get clock freq\n", thisfunc);

  if (verbose)
    printf ("GPTL: %s: Clock rate = %f MHz\n", thisfunc, cpumhz);

  cyc2sec = 1./(cpumhz * 1.e6);
  return 0;
#else
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

__host__ static inline double utr_nanotime ()
{
#ifdef HAVE_NANOTIME
  double timestamp;
  timestamp = nanotime () * cyc2sec;
  return timestamp;
#else
  static const char *thisfunc = "utr_nanotime";
  (void) GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
  return -1.;
#endif
}

/*
** MPI_Wtime requires MPI lib.
*/
__host__ static int init_mpiwtime ()
{
#ifdef HAVE_MPI
  return 0;
#else
  static const char *thisfunc = "init_mpiwtime";
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

__host__ static inline double utr_mpiwtime ()
{
#ifdef HAVE_MPI
  return MPI_Wtime ();
#else
  static const char *thisfunc = "utr_mpiwtime";
  (void) GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
  return -1.;
#endif
}

/* 
** Probably need to link with -lrt for this one to work 
*/
__host__ static int init_clock_gettime ()
{
  static const char *thisfunc = "init_clock_gettime";
#ifdef HAVE_LIBRT
  struct timespec tp;
  (void) clock_gettime (CLOCK_REALTIME, &tp);
  ref_clock_gettime = tp.tv_sec;
  if (verbose)
    printf ("GPTL: %s: ref_clock_gettime=%ld\n", thisfunc, (long) ref_clock_gettime);
  return 0;
#else
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

__host__ static inline double utr_clock_gettime ()
{
#ifdef HAVE_LIBRT
  struct timespec tp;
  (void) clock_gettime (CLOCK_REALTIME, &tp);
  return (tp.tv_sec - ref_clock_gettime) + 1.e-9*tp.tv_nsec;
#else
  static const char *thisfunc = "utr_clock_gettime";
  (void) GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
  return -1.;
#endif
}

/*
** High-res timer on AIX: read_real_time
*/
__host__ static int init_read_real_time ()
{
  static const char *thisfunc = "init_read_real_time";
#ifdef _AIX
  timebasestruct_t ibmtime;
  (void) read_real_time (&ibmtime, TIMEBASE_SZ);
  (void) time_base_to_time (&ibmtime, TIMEBASE_SZ);
  ref_read_real_time = ibmtime.tb_high;
  if (verbose)
    printf ("GPTL: %s: ref_read_real_time=%ld\n", thisfunc, (long) ref_read_real_time);
  return 0;
#else
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

__host__ static inline double utr_read_real_time ()
{
#ifdef _AIX
  timebasestruct_t ibmtime;
  (void) read_real_time (&ibmtime, TIMEBASE_SZ);
  (void) time_base_to_time (&ibmtime, TIMEBASE_SZ);
  return (ibmtime.tb_high - ref_read_real_time) + 1.e-9*ibmtime.tb_low;
#else
  static const char *thisfunc = "utr_read_real_time";
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

/*
** Default available most places: gettimeofday
*/
__host__ static int init_gettimeofday ()
{
  static const char *thisfunc = "init_gettimeofday";
#ifdef HAVE_GETTIMEOFDAY
  struct timeval tp;
  (void) gettimeofday (&tp, 0);
  ref_gettimeofday = tp.tv_sec;
  if (verbose)
    printf ("GPTL: %s: ref_gettimeofday=%ld\n", thisfunc, (long) ref_gettimeofday);
  return 0;
#else
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

__host__ static inline double utr_gettimeofday ()
{
#ifdef HAVE_GETTIMEOFDAY
  struct timeval tp;
  (void) gettimeofday (&tp, 0);
  return (tp.tv_sec - ref_gettimeofday) + 1.e-6*tp.tv_usec;
#else
  static const char *thisfunc = "utr_gettimeofday";
  return GPTLerror ("GPTL: %s: not enabled\n", thisfunc);
#endif
}

/*
** placebo: does nothing and returns zero always. Useful for estimating overhead costs
*/
__host__ static int init_placebo ()
{
  return 0;
}

__host__ static inline double utr_placebo ()
{
  static const double zero = 0.;
  return zero;
}

/*
** printself_andchildren: Recurse through call tree, printing stats for self, then children
*/
__host__ static void printself_andchildren (const Timer *ptr,
					    FILE *fp, 
					    int t,
					    int depth,
					    double self_ohd,
					    double parent_ohd)
{
  int n;

  if (depth > -1)     /* -1 flag is to avoid printing stats for dummy outer timer */
    printstats (ptr, fp, t, depth, true, self_ohd, parent_ohd);

  for (n = 0; n < ptr->nchildren; n++)
    printself_andchildren (ptr->children[n], fp, t, depth+1, self_ohd, parent_ohd);
}

/*
** GPTLget_nthreads: Return number of threads. NOT a public entry point
*/
__host__ int GPTLget_nthreads ()
{
  return nthreads;
}

/*
** GPTLget_timersaddr: Return address of timers. NOT a public entry point
*/
__host__ Timer **GPTLget_timersaddr ()
{
  return timers;
}

// Return useful GPU properties. Use arg list for SMcount, cores_per_sm, and cores_per_gpu even 
// though they're globals, because this is a user-callable routine
__host__ int GPTLget_gpu_props (int *khz, int *warpsize, int *devnum, int *SMcount,
				int *cores_per_sm, int *cores_per_gpu)
{
  hipDeviceProp_t prop;
  size_t size;
  hipError_t err;
  static const size_t onemb = 1024 * 1024;
  //static const size_t heap_mb = 8;  // this number should avoid needing to reset the limit
  //static const size_t heap_mb = 128;
  static const char *thisfunc = "GPTLget_gpu_props";

  if ((err = hipGetDeviceProperties (&prop, 0)) != hipSuccess) {
    printf ("%s: error:%s", thisfunc, hipGetErrorString (err));
    return -1;
  }

  *khz           = prop.clockRate;
  *warpsize      = prop.warpSize;
  *SMcount       = prop.multiProcessorCount;
  *cores_per_sm  = _ConvertSMVer2Cores (prop.major, prop.minor);
  *cores_per_gpu = *cores_per_sm * (*SMcount);
  
  // Use _ConvertSMVer2Cores when it is available from nvidia
  //  cores_per_gpu = _ConvertSMVer2Cores (prop.major, prop.minor) * prop.multiProcessorCount);
  printf ("%s: major.minor=%d.%d\n", thisfunc, prop.major, prop.minor);
  printf ("%s: SM count=%d\n",      thisfunc, *SMcount);
  printf ("%s: cores per sm=%d\n",  thisfunc, *cores_per_sm);
  printf ("%s: cores per GPU=%d\n", thisfunc, *cores_per_gpu);

  err = hipGetDevice (devnum);  // device number
  err = hipDeviceGetLimit (&size, hipLimitMallocHeapSize);
  printf ("%s: default hipLimitMallocHeapSize=%d MB\n", thisfunc, (int) (size / onemb));
  return 0;
}

__host__ int GPTLcompute_chunksize (const int oversub, const int inner_iter_count)
{
  int chunksize;
  float oversub_factor;
  static const char *thisfunc = "GPTLcompute_chunksize";

  if (oversub < 1)
    return GPTLerror ("%s: oversub=%d must be > 0\n", thisfunc, oversub);

  chunksize = (oversub * GPTLcores_per_gpu) / inner_iter_count;
  if (chunksize < 1) {
    chunksize = 1;
    oversub_factor = (float) inner_iter_count / (float) GPTLcores_per_gpu;
    printf ("%s: WARNING: chunksize=1 still results in an oversubscription factor=%f compared to request=%d\n",
	    thisfunc, oversub_factor, oversub);
  }
  return chunksize;
}

__host__ int GPTLcudadevsync (void)
{
  hipDeviceSynchronize ();
  return 0;
}

/*************************************************************************************/

/*
** Contents of inserted threadutil.c starts here.
** Moved to gptl.c to enable inlining
*/

/*
**
** Author: Jim Rosinski
** 
** Utility functions handle thread-based GPTL needs.
*/

/**********************************************************************************/
/* 
** 2 sets of routines: OMP threading, unthreaded
*/

#if ( defined THREADED_OMP )

/*
** threadinit: Allocate and initialize GPTLthreadid_omp; set max number of threads
**
** Output results:
**   maxthreads: max number of threads
**
**   GPTLthreadid_omp[] is allocated and initialized to -1
**
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__host__ static int threadinit (void)
{
  int t;  /* loop index */
  static const char *thisfunc = "threadinit";

  if (omp_get_thread_num () != 0)
    return GPTLerror ("OMP %s: MUST only be called by the master thread\n", thisfunc);

  /* 
  ** Allocate the threadid array which maps physical thread IDs to logical IDs 
  ** For OpenMP this will be just GPTLthreadid_omp[iam] = iam;
  */
  if (GPTLthreadid_omp) 
    return GPTLerror ("OMP %s: has already been called.\nMaybe mistakenly called by multiple threads?", 
                      thisfunc);

  /*
  ** maxthreads may have been set by the user, in which case use that. But if as 
  ** yet uninitialized, set to the current value of OMP_NUM_THREADS. 
  */
  if (maxthreads == -1)
    maxthreads = MAX ((1), (omp_get_max_threads ()));

  if ( ! (GPTLthreadid_omp = (int *) GPTLallocate (maxthreads * sizeof (int), thisfunc)))
    return GPTLerror ("OMP %s: malloc failure for %d elements of GPTLthreadid_omp\n", thisfunc, maxthreads);

  /*
  ** Initialize threadid array to flag values for use by get_thread_num().
  ** get_thread_num() will fill in the values on first use.
  */
  for (t = 0; t < maxthreads; ++t)
    GPTLthreadid_omp[t] = -1;

#ifdef VERBOSE
  printf ("GPTL: OMP %s: Set maxthreads=%d\n", thisfunc, maxthreads);
#endif
  
  return 0;
}

/*
** Threadfinalize: clean up
**
** Output results:
**   GPTLthreadid_omp array is freed and array pointer nullified
*/
__host__ static void threadfinalize ()
{
  free ((void *) GPTLthreadid_omp);
  GPTLthreadid_omp = 0;
}

/*
** get_thread_num: Determine thread number of the calling thread
**
** Output results:
**   nthreads:     Number of threads
**   GPTLthreadid_omp: Our thread id added to list on 1st call
**
** Return value: thread number (success) or GPTLerror (failure)
**   5/8/16: Modified to enable 2-level OMP nesting: Fold combination of current and parent
**   thread info into a single index
*/
__host__ static inline int get_thread_num (void)
{
  int t;        /* thread number */
  static const char *thisfunc = "get_thread_num";

#ifdef ENABLE_NESTEDOMP
  int myid;            /* my thread id */
  int lvl;             /* nest level: Currently only 2 nesting levels supported */
  int parentid;        /* thread number of parent team */
  int my_nthreads;     /* number of threads in the parent team */

  myid = omp_get_thread_num ();
  if (omp_get_nested ()) {         /* nesting is "enabled", though not necessarily active */
    lvl = omp_get_active_level (); /* lvl=2 => inside 2 #pragma omp regions */
    if (lvl < 2) {
      /* 0 or 1-level deep: simply use thread id as index */
      t = myid;
    } else if (lvl == 2) {
      /* Create a unique id "t" for indexing into singly-dimensioned thread array */
      parentid    = omp_get_ancestor_thread_num (lvl-1);
      my_nthreads = omp_get_team_size (lvl);
      t           = parentid*my_nthreads + myid;
    } else {
      return GPTLerror ("OMP %s: GPTL supports only 2 nested OMP levels got %d\n", thisfunc, lvl);
    }
  } else {
    /* un-nested case: thread id is index */
    t = myid;
  }
#else
  t = omp_get_thread_num ();
#endif
  if (t >= maxthreads)
    return GPTLerror ("OMP %s: returned id=%d exceeds maxthreads=%d\n", thisfunc, t, maxthreads);

  /* If our thread number has already been set in the list, we are done */
  if (t == GPTLthreadid_omp[t])
    return t;

  /* 
  ** Thread id not found. Modify GPTLthreadid_omp with our ID
  ** Due to the setting of GPTLthreadid_omp, everything below here will only execute once per thread.
  */
  GPTLthreadid_omp[t] = t;

#ifdef VERBOSE
  printf ("GPTL: OMP %s: 1st call t=%d\n", thisfunc, t);
#endif

  /* nthreads = maxthreads based on setting in threadinit or user call to GPTLsetoption() */
  nthreads = maxthreads;
#ifdef VERBOSE
  printf ("GPTL: OMP %s: nthreads=%d\n", thisfunc, nthreads);
#endif

  return t;
}

/**********************************************************************************/
/*
** Unthreaded case
*/

#else

__host__ static int threadinit (void)
{
  static const char *thisfunc = "threadinit";

  if (nthreads != -1)
    return GPTLerror ("GPTL: Unthreaded %s: MUST only be called once", thisfunc);

  nthreads = 0;
  maxthreads = 1;
  return 0;
}

__host__ void threadfinalize ()
{
  GPTLthreadid = -1;
}

__host__ static inline int get_thread_num ()
{
  nthreads = 1;
  return 0;
}

#endif  /* Unthreaded case */
}
